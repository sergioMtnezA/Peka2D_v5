#include "hip/hip_runtime.h"
#include "memoryManage.cuh"

////////////////////////////////////////////////////////////////
EXPORT_DLL int createArraysCudaMemory(
    t_arrays *carrays,
    t_arrays *garrays,
    t_cuPtr *cuPtr,
    t_message *msg){
/*----------------------------*/

	int i,j;
    int NCwall;
	int ncells,nwc,nwb;
    int nWallCell;

	size_t free_mem, total_mem;

    int nTasks, blocksPerGrid;

	//Local variables just for allocation
    NCwall=carrays->NCwall;	//walls per cell
	ncells=carrays->ncells;
    nWallCell=NCwall*ncells;
	nwc=carrays->nw_calc;
	nwb=carrays->nw_bound;

    //allocate arrays memory
    if(!allocateArraysCudaMem( 
        NCwall, ncells, nWallCell, nwc, nwb, 
        cuPtr) ){
        return(0);
    };
    hipDeviceSynchronize();
    //getCudaMemoryState(&free_mem, &total_mem);   



    //allocate  control arrays memory
    if(!copyControlArraysCudaMem(carrays, garrays, cuPtr)){
        return(0);
    };
    hipDeviceSynchronize();
    //getCudaMemoryState(&free_mem, &total_mem); 


    //copy mesh arrays memory
    if(!copyMeshArraysCudaMem(carrays, garrays, cuPtr)){
        return(0);
    };
    hipDeviceSynchronize();
    //getCudaMemoryState(&free_mem, &total_mem);     


    //assign CUDA garrays to CUDA cuPtr 
    assignGArraysToCudaMem <<<1,1>>> (garrays,
        //------------------------cells
        cuPtr->activeC,
        cuPtr->actCells,
        cuPtr->cidx,
        cuPtr->nneig,
        cuPtr->z,
        cuPtr->h,
        cuPtr->hu,
        cuPtr->hv,
        cuPtr->u,
        cuPtr->v,
        cuPtr->modulou,
        cuPtr->sqrh,
        cuPtr->area,
        cuPtr->nman,
        cuPtr->SOX,
        cuPtr->SOY,
        cuPtr->mass,
        //--------------------- cells*NCwall
        cuPtr->dh,
        cuPtr->dhu,
        cuPtr->dhv,
        cuPtr->solidWallByCell,
        cuPtr->neighCell,
        cuPtr->neighWall,
        cuPtr->normalXbyCell,
        cuPtr->normalYbyCell,
        //---------------------- internal walls
        cuPtr->activeW,
        cuPtr->actWalls,
        cuPtr->widx,
        cuPtr->idx1,
        cuPtr->idx2,
        cuPtr->idw1,
        cuPtr->idw2,
        cuPtr->normalX,
        cuPtr->normalY,
        cuPtr->deltaX,
        cuPtr->length,
        cuPtr->distNormal,
        cuPtr->distCentX,
        cuPtr->distCentY,
        cuPtr->nman2wall,
        cuPtr->gp,
        cuPtr->typeOfBound,
        cuPtr->solidWall,
        cuPtr->qnormalL,
        cuPtr->localDt
    );
    hipDeviceSynchronize();
    getCudaMemoryState(&free_mem, &total_mem); 

    return 1;
}




////////////////////////////////////////////////////////////////
EXPORT_DLL int allocateArraysCudaMem(
    int NCwall, int ncells, int nWallCell, int nwc, int nwb,
    t_cuPtr *cuPtr){
/*----------------------------*/

    //computation controls
    hipMalloc((void**) &(cuPtr->index), sizeof(int));
    hipMalloc((void**) &(cuPtr->check), sizeof(int));

    hipMalloc((void**) &(cuPtr->t), sizeof(double));
    hipMalloc((void**) &(cuPtr->dt), sizeof(double));

    hipMalloc((void**) &(cuPtr->nActCells), sizeof(int));
    hipMalloc((void**) &(cuPtr->nActWalls), sizeof(int));

    hipMalloc((void**) &(cuPtr->nIter), sizeof(int));
    hipMalloc((void**) &(cuPtr->indexOut), sizeof(int));
    hipMalloc((void**) &(cuPtr->indexDump), sizeof(int));
    hipMalloc((void**) &(cuPtr->dumpComponent), sizeof(int));
    hipMalloc((void**) &(cuPtr->dumpState), sizeof(int));

    hipMalloc((void**) &(cuPtr->massOld), sizeof(double));
    hipMalloc((void**) &(cuPtr->massNew), sizeof(double));
    hipMalloc((void**) &(cuPtr->massError), sizeof(double));

    hipMalloc((void**) &(cuPtr->qTotalIn), sizeof(double));
    hipMalloc((void**) &(cuPtr->qTotalOut), sizeof(double));


    //cells
    hipMalloc((void**) &(cuPtr->activeC), ncells*sizeof(int));
    hipMalloc((void**) &(cuPtr->actCells), ncells*sizeof(int));

    hipMalloc((void**) &(cuPtr->cidx), ncells*sizeof(int));
    hipMalloc((void**) &(cuPtr->nneig), ncells*sizeof(int));

    hipMalloc((void**) &(cuPtr->z), ncells*sizeof(double));
    hipMalloc((void**) &(cuPtr->h), ncells*sizeof(double));
    hipMalloc((void**) &(cuPtr->hu), ncells*sizeof(double));
    hipMalloc((void**) &(cuPtr->hv), ncells*sizeof(double));
    hipMalloc((void**) &(cuPtr->u), ncells*sizeof(double));
    hipMalloc((void**) &(cuPtr->v), ncells*sizeof(double));
    hipMalloc((void**) &(cuPtr->modulou), ncells*sizeof(double));
    hipMalloc((void**) &(cuPtr->sqrh), ncells*sizeof(double));

    hipMalloc((void**) &(cuPtr->area), ncells*sizeof(double));
    hipMalloc((void**) &(cuPtr->nman), ncells*sizeof(double));
    hipMalloc((void**) &(cuPtr->SOX), ncells*sizeof(double));
    hipMalloc((void**) &(cuPtr->SOY), ncells*sizeof(double));    

    hipMalloc((void**) &(cuPtr->mass), ncells*sizeof(double));


    //cells*NCwall
    hipMalloc((void**) &(cuPtr->dh), nWallCell*sizeof(double));
    hipMalloc((void**) &(cuPtr->dhu), nWallCell*sizeof(double));
    hipMalloc((void**) &(cuPtr->dhv), nWallCell*sizeof(double));

    hipMalloc((void**) &(cuPtr->solidWallByCell), nWallCell*sizeof(int));
    hipMalloc((void**) &(cuPtr->neighCell), nWallCell*sizeof(int));
    hipMalloc((void**) &(cuPtr->neighWall), nWallCell*sizeof(int));

    hipMalloc((void**) &(cuPtr->normalXbyCell), nWallCell*sizeof(double));
    hipMalloc((void**) &(cuPtr->normalYbyCell), nWallCell*sizeof(double));


    //internal walls
    hipMalloc((void**) &(cuPtr->activeW), nwc*sizeof(int));
    hipMalloc((void**) &(cuPtr->actWalls), nwc*sizeof(int));

    hipMalloc((void**) &(cuPtr->widx), nwc*sizeof(int));

    hipMalloc((void**) &(cuPtr->idx1), nwc*sizeof(int));
    hipMalloc((void**) &(cuPtr->idx2), nwc*sizeof(int));
    hipMalloc((void**) &(cuPtr->idw1), nwc*sizeof(int));
    hipMalloc((void**) &(cuPtr->idw2), nwc*sizeof(int)); 

    hipMalloc((void**) &(cuPtr->normalX), nwc*sizeof(double));
    hipMalloc((void**) &(cuPtr->normalY), nwc*sizeof(double));
    hipMalloc((void**) &(cuPtr->deltaX), nwc*sizeof(double));
    hipMalloc((void**) &(cuPtr->length), nwc*sizeof(double));
    hipMalloc((void**) &(cuPtr->distNormal), nwc*sizeof(double));
    hipMalloc((void**) &(cuPtr->distCentX), nwc*sizeof(double));
    hipMalloc((void**) &(cuPtr->distCentY), nwc*sizeof(double)); 

    hipMalloc((void**) &(cuPtr->nman2wall), nwc*sizeof(double));
    hipMalloc((void**) &(cuPtr->gp), nwc*sizeof(double));

    hipMalloc((void**) &(cuPtr->typeOfBound), nwc*sizeof(int));
    hipMalloc((void**) &(cuPtr->solidWall), nwc*sizeof(int));

    hipMalloc((void**) &(cuPtr->qnormalL), nwc*sizeof(double));
    hipMalloc((void**) &(cuPtr->localDt), nwc*sizeof(double));


    //boundaries

    
      
    return(1);

}



////////////////////////////////////////////////////////////////
EXPORT_DLL int copyControlArraysCudaMem(
    t_arrays *carrays,
    t_arrays *garrays,
    t_cuPtr *cuPtr){
/*----------------------------*/

    int iaux=0;

    // COMPUTATION CONTROLS ////////////////////////////////
    hipMemcpy(cuPtr->index, &(iaux), sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy(cuPtr->check, &(iaux), sizeof(int), hipMemcpyHostToDevice );

    hipMemcpy(cuPtr->t , &(carrays->t), sizeof(double), hipMemcpyHostToDevice );
    hipMemcpy(cuPtr->dt, &(carrays->dt), sizeof(double), hipMemcpyHostToDevice );

    hipMemcpy(cuPtr->nActCells, &(carrays->nActCells), sizeof(int), hipMemcpyHostToDevice ); //initial active cells
    hipMemcpy(cuPtr->nActWalls, &(carrays->nActWalls), sizeof(int), hipMemcpyHostToDevice ); //initial active walls

    hipMemcpy(cuPtr->nIter, &(carrays->nIter), sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy(cuPtr->indexOut, &(carrays->indexOut), sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy(cuPtr->indexDump, &(carrays->indexDump), sizeof(int), hipMemcpyHostToDevice ); 
    hipMemcpy(cuPtr->dumpComponent, &(carrays->dumpComponent), sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy(cuPtr->dumpState, &(carrays->dumpState), sizeof(int), hipMemcpyHostToDevice );

    hipMemcpy(cuPtr->massOld, &(carrays->massOld), sizeof(double), hipMemcpyHostToDevice );
    hipMemcpy(cuPtr->massNew, &(carrays->massNew), sizeof(double), hipMemcpyHostToDevice );
    hipMemcpy(cuPtr->massError, &(carrays->massError), sizeof(double), hipMemcpyHostToDevice );

    hipMemcpy(cuPtr->qTotalIn, &(carrays->qTotalIn), sizeof(double), hipMemcpyHostToDevice );
    hipMemcpy(cuPtr->qTotalOut, &(carrays->qTotalOut), sizeof(double), hipMemcpyHostToDevice );




    // COMPUTATION ARRAYS ////////////////////////////////
    hipMemcpy(&(garrays->ncores), &(carrays->ncores), sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy(&(garrays->gpuid), &(carrays->gpuid), sizeof(int), hipMemcpyHostToDevice );

    hipMemcpy(&(garrays->ti), &(carrays->ti), sizeof(double), hipMemcpyHostToDevice );
    hipMemcpy(&(garrays->tf), &(carrays->tf), sizeof(double), hipMemcpyHostToDevice );
    hipMemcpy(&(garrays->CFL), &(carrays->CFL), sizeof(double), hipMemcpyHostToDevice );

    hipMemcpy(&(garrays->writeMass), &(carrays->writeMass), sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy(&(garrays->writeExtremes), &(carrays->writeExtremes), sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy(&(garrays->indexWriteHotstart), &(carrays->indexWriteHotstart), sizeof(int), hipMemcpyHostToDevice );

    hipMemcpy(&(garrays->nIterOut), &(carrays->nIterOut), sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy(&(garrays->dtOut), &(carrays->dtOut), sizeof(double), hipMemcpyHostToDevice );
    hipMemcpy(&(garrays->dtDump), &(carrays->dtDump), sizeof(double), hipMemcpyHostToDevice );
    hipMemcpy(&(garrays->minh), &(carrays->minh), sizeof(double), hipMemcpyHostToDevice );

    //mesh
    hipMemcpy(&(garrays->NCwall), &(carrays->NCwall), sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy(&(garrays->ncells), &(carrays->ncells), sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy(&(garrays->nWallCell), &(carrays->nWallCell), sizeof(int), hipMemcpyHostToDevice );

    hipMemcpy(&(garrays->nw_calc), &(carrays->nw_calc), sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy(&(garrays->nw_bound), &(carrays->nw_bound), sizeof(int), hipMemcpyHostToDevice );

    hipMemcpy(&(garrays->nInlet), &(carrays->nInlet), sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy(&(garrays->nOutlet), &(carrays->nOutlet), sizeof(int), hipMemcpyHostToDevice );

    //computation controls
    hipMemcpy(&(garrays->t), &(carrays->t), sizeof(double), hipMemcpyHostToDevice );
    hipMemcpy(&(garrays->dt), &(carrays->dt), sizeof(double), hipMemcpyHostToDevice );

    hipMemcpy(&(garrays->nActCells), &(carrays->nActCells), sizeof(int), hipMemcpyHostToDevice ); //initial active cells
    hipMemcpy(&(garrays->nActWalls), &(carrays->nActWalls), sizeof(int), hipMemcpyHostToDevice ); //initial active walls

    hipMemcpy(&(garrays->nIter), &(carrays->nIter), sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy(&(garrays->indexOut), &(carrays->indexOut), sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy(&(garrays->indexDump), &(carrays->indexDump), sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy(&(garrays->dumpComponent), &(carrays->dumpComponent), sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy(&(garrays->dumpState), &(carrays->dumpState), sizeof(int), hipMemcpyHostToDevice );

    hipMemcpy(&(garrays->massOld), &(carrays->massOld), sizeof(double), hipMemcpyHostToDevice );
    hipMemcpy(&(garrays->massNew), &(carrays->massNew), sizeof(double), hipMemcpyHostToDevice );
    hipMemcpy(&(garrays->massError), &(carrays->massError), sizeof(double), hipMemcpyHostToDevice );

    //boundary control
    hipMemcpy(&(garrays->qTotalIn), &(carrays->qTotalIn), sizeof(double), hipMemcpyHostToDevice );
    hipMemcpy(&(garrays->qTotalOut), &(carrays->qTotalOut), sizeof(double), hipMemcpyHostToDevice );


    return 1;

}



////////////////////////////////////////////////////////////////
int copyMeshArraysCudaMem(
    t_arrays *carrays,
    t_arrays *garrays,
    t_cuPtr *cuPtr){
/*----------------------------*/
    
	int i,j;
    int NCwall;
	int ncells,nwc,nwb;
    int nWallCell;

	//Local variables just for allocation
    NCwall=carrays->NCwall;	//walls per cell
	ncells=carrays->ncells;
    nWallCell=NCwall*ncells;
	nwc=carrays->nw_calc;
	nwb=carrays->nw_bound;

    // COMPUTATION ARRAYS ////////////////////////////////////////////
    //cells    
    hipMemcpy((cuPtr->activeC), (carrays->activeC), ncells*sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy((cuPtr->actCells), (carrays->actCells), ncells*sizeof(int), hipMemcpyHostToDevice );    

    hipMemcpy((cuPtr->cidx), (carrays->cidx), ncells*sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy((cuPtr->nneig), (carrays->nneig), ncells*sizeof(int), hipMemcpyHostToDevice );

    hipMemcpy((cuPtr->z), (carrays->z), ncells*sizeof(double), hipMemcpyHostToDevice );
    hipMemcpy((cuPtr->h), (carrays->h), ncells*sizeof(double), hipMemcpyHostToDevice );
    hipMemcpy((cuPtr->hu), (carrays->hu), ncells*sizeof(double), hipMemcpyHostToDevice );
    hipMemcpy((cuPtr->hv), (carrays->hv), ncells*sizeof(double), hipMemcpyHostToDevice );
    hipMemcpy((cuPtr->u), (carrays->u), ncells*sizeof(double), hipMemcpyHostToDevice );
    hipMemcpy((cuPtr->v), (carrays->v), ncells*sizeof(double), hipMemcpyHostToDevice );
    hipMemcpy((cuPtr->modulou), (carrays->modulou), ncells*sizeof(double), hipMemcpyHostToDevice );
    hipMemcpy((cuPtr->sqrh), (carrays->sqrh), ncells*sizeof(double), hipMemcpyHostToDevice );
	
    hipMemcpy((cuPtr->area), (carrays->area), ncells*sizeof(double), hipMemcpyHostToDevice );
    hipMemcpy((cuPtr->nman), (carrays->nman), ncells*sizeof(double), hipMemcpyHostToDevice );
    hipMemcpy((cuPtr->SOX), (carrays->SOX), ncells*sizeof(double), hipMemcpyHostToDevice );
    hipMemcpy((cuPtr->SOY), (carrays->SOY), ncells*sizeof(double), hipMemcpyHostToDevice );    

    hipMemcpy((cuPtr->mass), (carrays->mass), ncells*sizeof(double), hipMemcpyHostToDevice );


    //cells*NCwall
    hipMemcpy((cuPtr->dh), (carrays->dh), nWallCell*sizeof(double), hipMemcpyHostToDevice );
    hipMemcpy((cuPtr->dhu), (carrays->dhu), nWallCell*sizeof(double), hipMemcpyHostToDevice );
    hipMemcpy((cuPtr->dhv), (carrays->dhv), nWallCell*sizeof(double), hipMemcpyHostToDevice );

    hipMemcpy((cuPtr->solidWallByCell), (carrays->solidWallByCell), nWallCell*sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy((cuPtr->neighCell), (carrays->neighCell), nWallCell*sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy((cuPtr->neighWall), (carrays->neighWall), nWallCell*sizeof(int), hipMemcpyHostToDevice );

    hipMemcpy((cuPtr->normalXbyCell), (carrays->normalXbyCell), nWallCell*sizeof(double), hipMemcpyHostToDevice );
    hipMemcpy((cuPtr->normalYbyCell), (carrays->normalYbyCell), nWallCell*sizeof(double), hipMemcpyHostToDevice );    


    //internal walls
    hipMemcpy((cuPtr->activeW), (carrays->activeW), nwc*sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy((cuPtr->actWalls), (carrays->actWalls), nwc*sizeof(int), hipMemcpyHostToDevice );

    hipMemcpy((cuPtr->widx), (carrays->widx), nwc*sizeof(int), hipMemcpyHostToDevice );

    hipMemcpy((cuPtr->idx1), (carrays->idx1), nwc*sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy((cuPtr->idx2), (carrays->idx2), nwc*sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy((cuPtr->idw1), (carrays->idw1), nwc*sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy((cuPtr->idw2), (carrays->idw2), nwc*sizeof(int), hipMemcpyHostToDevice );

    hipMemcpy((cuPtr->normalX), (carrays->normalX), nwc*sizeof(double), hipMemcpyHostToDevice );
    hipMemcpy((cuPtr->normalY), (carrays->normalY), nwc*sizeof(double), hipMemcpyHostToDevice ); 
    hipMemcpy((cuPtr->deltaX), (carrays->deltaX), nwc*sizeof(double), hipMemcpyHostToDevice ); 
    hipMemcpy((cuPtr->length), (carrays->length), nwc*sizeof(double), hipMemcpyHostToDevice ); 
    hipMemcpy((cuPtr->distNormal), (carrays->distNormal), nwc*sizeof(double), hipMemcpyHostToDevice ); 
    hipMemcpy((cuPtr->distCentX), (carrays->distCentX), nwc*sizeof(double), hipMemcpyHostToDevice ); 
    hipMemcpy((cuPtr->distCentY), (carrays->distCentY), nwc*sizeof(double), hipMemcpyHostToDevice ); 

    hipMemcpy((cuPtr->nman2wall), (carrays->nman2wall), nwc*sizeof(double), hipMemcpyHostToDevice ); 
    hipMemcpy((cuPtr->gp), (carrays->gp), nwc*sizeof(double), hipMemcpyHostToDevice ); 

    hipMemcpy((cuPtr->typeOfBound), (carrays->typeOfBound), nwc*sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy((cuPtr->solidWall), (carrays->solidWall), nwc*sizeof(int), hipMemcpyHostToDevice );
 
    hipMemcpy((cuPtr->qnormalL), (carrays->qnormalL), nwc*sizeof(double), hipMemcpyHostToDevice ); 
    hipMemcpy((cuPtr->localDt), (carrays->localDt), nwc*sizeof(double), hipMemcpyHostToDevice ); 

    //boundaries
  

    return(1);
    
}



////////////////////////////////////////////////////////////////
__global__ void assignGArraysToCudaMem(t_arrays *garrays,
	//------------------------cells
	int *activeC,
	int *actCells,
	int *cidx,
	int *nneig,
	double *z,
	double *h,
	double *hu,
	double *hv,
	double *u,
	double *v,
	double *modulou,
	double *sqrh,
	double *area,
	double *nman,
	double *SOX,
	double *SOY,
	double *mass,
	//--------------------- cells*NCwall
	double *dh,
	double *dhu,
	double *dhv,
	int *solidWallByCell,
	int *neighCell,
	int *neighWall,
	double *normalXbyCell,
	double *normalYbyCell,
	//---------------------- internal walls
	int *activeW,
	int *actWalls,
	int *widx,
	int *idx1,
	int *idx2,
	int *idw1,
	int *idw2,
	double *normalX,
	double *normalY,
	double *deltaX,
	double *length,
	double *distNormal,
	double *distCentX,
	double *distCentY,
	double *nman2wall,
	double *gp,
	int *typeOfBound,
	int *solidWall,
	double *qnormalL,
	double *localDt){
/*----------------------------*/
	//mesh
	garrays->activeC=activeC;
	garrays->actCells=actCells;

	garrays->cidx=cidx;
	garrays->nneig=nneig;

	garrays->z=z;
	garrays->h=h;
	garrays->hu=hu;
	garrays->hv=hv;
	garrays->u=u;
	garrays->v=v;
	garrays->modulou=modulou;
	garrays->sqrh=sqrh;

	garrays->area=area;
	garrays->nman=nman;
	garrays->SOX=SOX;
	garrays->SOY=SOY;

	garrays->mass=mass;

	//cells*NCwall
	garrays->dh=dh;
	garrays->dhu=dhu;
	garrays->dhv=dhv;
	garrays->solidWallByCell=solidWallByCell;
	garrays->neighCell=neighCell;
	garrays->neighWall=neighWall;
	garrays->normalXbyCell=normalXbyCell;
	garrays->normalYbyCell=normalYbyCell;

	//internal walls
	garrays->activeW=activeW;
	garrays->actWalls=actWalls;

	garrays->widx=widx;

	garrays->idx1=idx1;
	garrays->idx2=idx2;
	garrays->idw1=idw1;
	garrays->idw2=idw2;

	garrays->normalX=normalX;
	garrays->normalY=normalY;
	garrays->deltaX=deltaX;
	garrays->length=length;
	garrays->distNormal=distNormal;
	garrays->distCentX=distCentX;
	garrays->distCentY=distCentY;

	garrays->nman2wall=nman2wall;
	garrays->gp=gp;

	garrays->typeOfBound=typeOfBound;
	garrays->solidWall=solidWall;

	garrays->qnormalL=qnormalL;
	garrays->localDt=localDt;

}




////////////////////////////////////////////////////////////////
EXPORT_DLL int freeCudaMemory(t_cuPtr *cuPtr){
/*----------------------------*/

    //computation controls
    hipFree(cuPtr->index);
    hipFree(cuPtr->check);

    hipFree(cuPtr->t);
    hipFree(cuPtr->dt);

    hipFree(cuPtr->nActCells);
    hipFree(cuPtr->nActWalls);

    hipFree(cuPtr->nIter);
    hipFree(cuPtr->indexOut);
    hipFree(cuPtr->indexDump);
    hipFree(cuPtr->dumpComponent);
    hipFree(cuPtr->dumpState);

    hipFree(cuPtr->massOld);
    hipFree(cuPtr->massNew);
    hipFree(cuPtr->massError);

    hipFree(cuPtr->qTotalIn);
    hipFree(cuPtr->qTotalOut);
    


    //cells
    hipFree(cuPtr->activeC);
    hipFree(cuPtr->actCells);

    hipFree(cuPtr->cidx);
    hipFree(cuPtr->nneig);

    hipFree(cuPtr->z);
    hipFree(cuPtr->h);
    hipFree(cuPtr->hu);
    hipFree(cuPtr->hv);
    hipFree(cuPtr->u);
    hipFree(cuPtr->v);
    hipFree(cuPtr->modulou);
    hipFree(cuPtr->sqrh);

    hipFree(cuPtr->area);
    hipFree(cuPtr->nman);
    hipFree(cuPtr->SOX);
    hipFree(cuPtr->SOY);  

    hipFree(cuPtr->mass);


    //cells*NCwall
    hipFree(cuPtr->dh);
    hipFree(cuPtr->dhu);
    hipFree(cuPtr->dhv);

    hipFree(cuPtr->solidWallByCell);
    hipFree(cuPtr->neighCell);
    hipFree(cuPtr->neighWall);

    hipFree(cuPtr->normalXbyCell);
    hipFree(cuPtr->normalYbyCell);


    //internal walls
    hipFree(cuPtr->activeW);
    hipFree(cuPtr->actWalls);

    hipFree(cuPtr->widx);

    hipFree(cuPtr->idx1);
    hipFree(cuPtr->idx2);
    hipFree(cuPtr->idw1);
    hipFree(cuPtr->idw2);

    hipFree(cuPtr->normalX);
    hipFree(cuPtr->normalY);
    hipFree(cuPtr->deltaX);
    hipFree(cuPtr->length);
    hipFree(cuPtr->distNormal);
    hipFree(cuPtr->distCentX);
    hipFree(cuPtr->distCentY);

    hipFree(cuPtr->nman2wall);
    hipFree(cuPtr->gp);

    hipFree(cuPtr->typeOfBound);
    hipFree(cuPtr->solidWall);

    hipFree(cuPtr->qnormalL);
    hipFree(cuPtr->localDt);


    //boundaries

    
      
    return(1);

}

