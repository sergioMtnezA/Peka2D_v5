#include "hip/hip_runtime.h"
#include "memoryManage.cuh"

////////////////////////////////////////////////////////////////
EXPORT_DLL int createArraysCudaMemory(
    t_arrays *carrays,
    t_arrays *garrays,
    t_cuPtr *cuPtr,
    t_message *msg){
/*----------------------------*/

	int i,j;
    int NCwall;
	int ncells,nwc,nwb;
    int nWallCell;
    int nSolutes;

    int nInlet, nOutlet, nOBC;
    int nTotalBoundCells, nTotalInnerCells;
    int nTotalPointSeries;

	size_t free_mem, total_mem;

    int nTasks, blocksPerGrid;

	//Mesh variables for allocation
    NCwall=carrays->NCwall;	//walls per cell
	ncells=carrays->ncells;
    nWallCell=NCwall*ncells;
	nwc=carrays->nw_calc;
	nwb=carrays->nw_bound;
   
    //Bound variables for allocation
    nInlet = carrays->nInlet;
    nOutlet = carrays->nOutlet;
    nOBC = carrays->nOBC;
    nTotalBoundCells = carrays->nTotalBoundCells;
    nTotalInnerCells = carrays->nTotalInnerCells;
    nTotalPointSeries = carrays->nTotalPointSeries;

    //Solute variables for allocation
    nSolutes=carrays->nSolutes;

    //Transfer computation arrays 
    if(!copyComputationControls(
        carrays, 
        garrays, 
        cuPtr)
    ) return 0;

    if(!allocateArraysCudaMem( 
        NCwall, ncells, nWallCell, nwc, nwb, 
        cuPtr) 
    ) return 0;

    if(!copyMeshArraysCudaMem(
        carrays, 
        garrays, 
        cuPtr)
    ) return 0;      

    assignMeshArraysToCudaMem <<<1,1>>> (garrays,
        //------------------------cells
        cuPtr->activeC,
        cuPtr->actCells,
        cuPtr->cidx,
        cuPtr->nneig,
        cuPtr->z,
        cuPtr->h,
        cuPtr->hu,
        cuPtr->hv,
        cuPtr->u,
        cuPtr->v,
        cuPtr->modulou,
        cuPtr->sqrh,
        cuPtr->area,
        cuPtr->nman,
        cuPtr->SOX,
        cuPtr->SOY,
        cuPtr->mass,
        //--------------------- cells*NCwall
        cuPtr->dh,
        cuPtr->dhu,
        cuPtr->dhv,
        cuPtr->solidWallByCell,
        cuPtr->neighCell,
        cuPtr->neighWall,
        cuPtr->typeWallByCell,
        cuPtr->normalXbyCell,
        cuPtr->normalYbyCell,
        //---------------------- internal walls
        cuPtr->activeW,
        cuPtr->actWalls,
        cuPtr->widx,
        cuPtr->idx1,
        cuPtr->idx2,
        cuPtr->idw1,
        cuPtr->idw2,
        cuPtr->normalX,
        cuPtr->normalY,
        cuPtr->deltaX,
        cuPtr->length,
        cuPtr->distNormal,
        cuPtr->distCentX,
        cuPtr->distCentY,
        cuPtr->nman2wall,
        cuPtr->gp,
        cuPtr->typeOfBound,
        cuPtr->solidWall,
        cuPtr->qnormalL,
        cuPtr->localDt
    ); 

    hipDeviceSynchronize();
    //getCudaMemoryState(&free_mem, &total_mem);        



    //Transfer boundary arrays 
    if(!allocateBoundArraysCudaMem( 
        nOBC, nInlet, nOutlet,
        nTotalBoundCells, nTotalInnerCells, 
        nTotalPointSeries,
        nSolutes,
        cuPtr) 
    ) return 0; 

    if(!copyBoundSetupArraysCudaMem(
        carrays, 
        garrays, 
        cuPtr)
    ) return 0;  

    if(!copyBoundMeshArraysCudaMem(
        carrays, 
        garrays, 
        cuPtr)
    ) return 0; 

    assignBoundArraysToCudaMem <<<1,1>>> (garrays,
        //------------------------ bound geometry
        cuPtr->nCellsOBC,
        cuPtr->iniIndexOBC,
        cuPtr->idBoundOBC,
        cuPtr->typeOBC,
        cuPtr->flagInitializeOBC,
        cuPtr->blockSectionOBC,
        cuPtr->normalXOBC,
        cuPtr->normalYOBC,
        cuPtr->totalLengthOBC,
        cuPtr->totalAreaOBC,
        cuPtr->cellZminOBC,
        cuPtr->nInnerCellsOBC,
        cuPtr->iniInnerIndexOBC,
        //----------------------- bound cells
        cuPtr->cidxBound,
        cuPtr->zCellBound,
        cuPtr->areaCellBound,        
        cuPtr->nxWallBound,
        cuPtr->nyWallBound,
        cuPtr->lWallBound,
        //----------------------- inner cells
        cuPtr->cidxInner,
        //----------------------- time series
        cuPtr->nPointsSeriesOBC,
        cuPtr->iniIndexSeriesOBC,
        cuPtr->tSeriesOBC,
        cuPtr->qSeriesOBC,
        cuPtr->hzSeriesOBC,
        cuPtr->frSeriesOBC,
        cuPtr->phiSeriesOBC,
        //----------------------- mass balance pointers
        cuPtr->qBoundByCell,
        cuPtr->mBoundByCell,
        cuPtr->mInnerByCell,
        cuPtr->qInByInlet,
        cuPtr->qOutByOutlet,
        cuPtr->mInByInlet, 
        cuPtr->mOutByOutlet
    );     

    hipDeviceSynchronize();
    //getCudaMemoryState(&free_mem, &total_mem);  


    #if SET_SOLUTE
    if(!allocateSoluteArraysCudaMem( 
        nSolutes,
        NCwall, ncells, nWallCell, nwc, nwb, 
        cuPtr)
    ) return 0;


    if(!copySoluteArraysCudaMem(
        nSolutes, 
        carrays, garrays, cuPtr)
    ) return 0;        

    assignSoluteArraysToCudaMem <<<1,1>>> (nSolutes, garrays,
        //------------------------solutes
        cuPtr->typeDiff,
        cuPtr->k_xx,
        cuPtr->k_yy,
        //------------------------solutes*cells
        cuPtr->hphi,
        cuPtr->phi,
        cuPtr->localDtd,
        cuPtr->BTcell,
        //------------------------solutes*cells*NCwall
        cuPtr->dhphi,
        cuPtr->Bwall
    );
    // hipDeviceSynchronize();
    // //getCudaMemoryState(&free_mem, &total_mem); 
    #endif

    //Chech final CUDA memory
    hipDeviceSynchronize();
    getCudaMemoryState(&free_mem, &total_mem); 

    return 1;
}






////////////////////////////////////////////////////////////////
EXPORT_DLL int copyComputationControls(
    t_arrays *carrays,
    t_arrays *garrays,
    t_cuPtr *cuPtr){
/*----------------------------*/

    int iaux=0;

    // CONTROL ARRAYS ////////////////////////////////
    //simulation
    hipMemcpy(&(garrays->ncores), &(carrays->ncores), sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy(&(garrays->gpuid), &(carrays->gpuid), sizeof(int), hipMemcpyHostToDevice );

    hipMemcpy(&(garrays->ti), &(carrays->ti), sizeof(double), hipMemcpyHostToDevice );
    hipMemcpy(&(garrays->tf), &(carrays->tf), sizeof(double), hipMemcpyHostToDevice );
    hipMemcpy(&(garrays->CFL), &(carrays->CFL), sizeof(double), hipMemcpyHostToDevice );

    hipMemcpy(&(garrays->writeMass), &(carrays->writeMass), sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy(&(garrays->writeExtremes), &(carrays->writeExtremes), sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy(&(garrays->indexWriteHotstart), &(carrays->indexWriteHotstart), sizeof(int), hipMemcpyHostToDevice );

    hipMemcpy(&(garrays->nIterOut), &(carrays->nIterOut), sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy(&(garrays->dtOut), &(carrays->dtOut), sizeof(double), hipMemcpyHostToDevice );
    hipMemcpy(&(garrays->dtDump), &(carrays->dtDump), sizeof(double), hipMemcpyHostToDevice );
    hipMemcpy(&(garrays->minh), &(carrays->minh), sizeof(double), hipMemcpyHostToDevice );

    //mesh 
    hipMemcpy(&(garrays->NCwall), &(carrays->NCwall), sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy(&(garrays->ncells), &(carrays->ncells), sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy(&(garrays->nWallCell), &(carrays->nWallCell), sizeof(int), hipMemcpyHostToDevice );

    hipMemcpy(&(garrays->nw_calc), &(carrays->nw_calc), sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy(&(garrays->nw_bound), &(carrays->nw_bound), sizeof(int), hipMemcpyHostToDevice );

    //execution 
    hipMemcpy(&(garrays->t), &(carrays->t), sizeof(double), hipMemcpyHostToDevice );
    hipMemcpy(&(garrays->dt), &(carrays->dt), sizeof(double), hipMemcpyHostToDevice );

    hipMemcpy(&(garrays->nActCells), &(carrays->nActCells), sizeof(int), hipMemcpyHostToDevice ); //initial active cells
    hipMemcpy(&(garrays->nActWalls), &(carrays->nActWalls), sizeof(int), hipMemcpyHostToDevice ); //initial active walls

    hipMemcpy(&(garrays->nIter), &(carrays->nIter), sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy(&(garrays->indexOut), &(carrays->indexOut), sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy(&(garrays->indexDump), &(carrays->indexDump), sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy(&(garrays->dumpComponent), &(carrays->dumpComponent), sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy(&(garrays->dumpState), &(carrays->dumpState), sizeof(int), hipMemcpyHostToDevice );

    hipMemcpy(&(garrays->massOld), &(carrays->massOld), sizeof(double), hipMemcpyHostToDevice );
    hipMemcpy(&(garrays->massNew), &(carrays->massNew), sizeof(double), hipMemcpyHostToDevice );
    hipMemcpy(&(garrays->massError), &(carrays->massError), sizeof(double), hipMemcpyHostToDevice );

    //boundary permanent flag
    hipMemcpy(&(garrays->nInlet), &(carrays->nInlet), sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy(&(garrays->nOutlet), &(carrays->nOutlet), sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy(&(garrays->nOBC), &(carrays->nOBC), sizeof(int), hipMemcpyHostToDevice );

    //solute permanent flag
    hipMemcpy(&(garrays->nSolutes), &(carrays->nSolutes), sizeof(int), hipMemcpyHostToDevice );



    // CONTROL POINTERS ////////////////////////////////
    hipMalloc((void**) &(cuPtr->index), sizeof(int));
    hipMalloc((void**) &(cuPtr->check), sizeof(int));
    hipMemcpy(cuPtr->index, &(iaux), sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy(cuPtr->check, &(iaux), sizeof(int), hipMemcpyHostToDevice );    

    hipMalloc((void**) &(cuPtr->t), sizeof(double));
    hipMalloc((void**) &(cuPtr->dt), sizeof(double));
    hipMemcpy(cuPtr->t , &(carrays->t), sizeof(double), hipMemcpyHostToDevice );
    hipMemcpy(cuPtr->dt, &(carrays->dt), sizeof(double), hipMemcpyHostToDevice );    

    hipMalloc((void**) &(cuPtr->nActCells), sizeof(int));
    hipMalloc((void**) &(cuPtr->nActWalls), sizeof(int));
    hipMemcpy(cuPtr->nActCells, &(carrays->nActCells), sizeof(int), hipMemcpyHostToDevice ); //initial active cells
    hipMemcpy(cuPtr->nActWalls, &(carrays->nActWalls), sizeof(int), hipMemcpyHostToDevice ); //initial active walls

    hipMalloc((void**) &(cuPtr->nIter), sizeof(int));
    hipMalloc((void**) &(cuPtr->indexOut), sizeof(int));
    hipMalloc((void**) &(cuPtr->indexDump), sizeof(int));
    hipMalloc((void**) &(cuPtr->dumpComponent), sizeof(int));
    hipMalloc((void**) &(cuPtr->dumpState), sizeof(int));
    hipMemcpy(cuPtr->nIter, &(carrays->nIter), sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy(cuPtr->indexOut, &(carrays->indexOut), sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy(cuPtr->indexDump, &(carrays->indexDump), sizeof(int), hipMemcpyHostToDevice ); 
    hipMemcpy(cuPtr->dumpComponent, &(carrays->dumpComponent), sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy(cuPtr->dumpState, &(carrays->dumpState), sizeof(int), hipMemcpyHostToDevice );    

    hipMalloc((void**) &(cuPtr->massOld), sizeof(double));
    hipMalloc((void**) &(cuPtr->massNew), sizeof(double));
    hipMalloc((void**) &(cuPtr->massError), sizeof(double));
    hipMemcpy(cuPtr->massOld, &(carrays->massOld), sizeof(double), hipMemcpyHostToDevice );
    hipMemcpy(cuPtr->massNew, &(carrays->massNew), sizeof(double), hipMemcpyHostToDevice );
    hipMemcpy(cuPtr->massError, &(carrays->massError), sizeof(double), hipMemcpyHostToDevice );    

    return 1;

}





////////////////////////////////////////////////////////////////
EXPORT_DLL int allocateArraysCudaMem(
    int NCwall, int ncells, int nWallCell, int nwc, int nwb,
    t_cuPtr *cuPtr){
/*----------------------------*/

    //cells
    hipMalloc((void**) &(cuPtr->activeC), ncells*sizeof(int));
    hipMalloc((void**) &(cuPtr->actCells), ncells*sizeof(int));

    hipMalloc((void**) &(cuPtr->cidx), ncells*sizeof(int));
    hipMalloc((void**) &(cuPtr->nneig), ncells*sizeof(int));

    hipMalloc((void**) &(cuPtr->z), ncells*sizeof(double));
    hipMalloc((void**) &(cuPtr->h), ncells*sizeof(double));
    hipMalloc((void**) &(cuPtr->hu), ncells*sizeof(double));
    hipMalloc((void**) &(cuPtr->hv), ncells*sizeof(double));
    hipMalloc((void**) &(cuPtr->u), ncells*sizeof(double));
    hipMalloc((void**) &(cuPtr->v), ncells*sizeof(double));
    hipMalloc((void**) &(cuPtr->modulou), ncells*sizeof(double));
    hipMalloc((void**) &(cuPtr->sqrh), ncells*sizeof(double));

    hipMalloc((void**) &(cuPtr->area), ncells*sizeof(double));
    hipMalloc((void**) &(cuPtr->nman), ncells*sizeof(double));
    hipMalloc((void**) &(cuPtr->SOX), ncells*sizeof(double));
    hipMalloc((void**) &(cuPtr->SOY), ncells*sizeof(double));    

    hipMalloc((void**) &(cuPtr->mass), ncells*sizeof(double));


    //cells*NCwall
    hipMalloc((void**) &(cuPtr->dh), nWallCell*sizeof(double));
    hipMalloc((void**) &(cuPtr->dhu), nWallCell*sizeof(double));
    hipMalloc((void**) &(cuPtr->dhv), nWallCell*sizeof(double));

    hipMalloc((void**) &(cuPtr->solidWallByCell), nWallCell*sizeof(int));
    hipMalloc((void**) &(cuPtr->neighCell), nWallCell*sizeof(int));
    hipMalloc((void**) &(cuPtr->neighWall), nWallCell*sizeof(int));
    hipMalloc((void**) &(cuPtr->typeWallByCell), nWallCell*sizeof(int));

    hipMalloc((void**) &(cuPtr->normalXbyCell), nWallCell*sizeof(double));
    hipMalloc((void**) &(cuPtr->normalYbyCell), nWallCell*sizeof(double));


    //internal walls
    hipMalloc((void**) &(cuPtr->activeW), nwc*sizeof(int));
    hipMalloc((void**) &(cuPtr->actWalls), nwc*sizeof(int));

    hipMalloc((void**) &(cuPtr->widx), nwc*sizeof(int));

    hipMalloc((void**) &(cuPtr->idx1), nwc*sizeof(int));
    hipMalloc((void**) &(cuPtr->idx2), nwc*sizeof(int));
    hipMalloc((void**) &(cuPtr->idw1), nwc*sizeof(int));
    hipMalloc((void**) &(cuPtr->idw2), nwc*sizeof(int)); 

    hipMalloc((void**) &(cuPtr->normalX), nwc*sizeof(double));
    hipMalloc((void**) &(cuPtr->normalY), nwc*sizeof(double));
    hipMalloc((void**) &(cuPtr->deltaX), nwc*sizeof(double));
    hipMalloc((void**) &(cuPtr->length), nwc*sizeof(double));
    hipMalloc((void**) &(cuPtr->distNormal), nwc*sizeof(double));
    hipMalloc((void**) &(cuPtr->distCentX), nwc*sizeof(double));
    hipMalloc((void**) &(cuPtr->distCentY), nwc*sizeof(double)); 

    hipMalloc((void**) &(cuPtr->nman2wall), nwc*sizeof(double));
    hipMalloc((void**) &(cuPtr->gp), nwc*sizeof(double));

    hipMalloc((void**) &(cuPtr->typeOfBound), nwc*sizeof(int));
    hipMalloc((void**) &(cuPtr->solidWall), nwc*sizeof(int));

    hipMalloc((void**) &(cuPtr->qnormalL), nwc*sizeof(double));
    hipMalloc((void**) &(cuPtr->localDt), nwc*sizeof(double));
      
    return(1);

}




////////////////////////////////////////////////////////////////
int copyMeshArraysCudaMem(
    t_arrays *carrays,
    t_arrays *garrays,
    t_cuPtr *cuPtr){
/*----------------------------*/
    
	int i,j;
    int NCwall;
	int ncells,nwc,nwb;
    int nWallCell;

	//Local variables just for allocation
    NCwall=carrays->NCwall;	//walls per cell
	ncells=carrays->ncells;
    nWallCell=NCwall*ncells;
	nwc=carrays->nw_calc;
	nwb=carrays->nw_bound;

    // COMPUTATION ARRAYS ////////////////////////////////////////////
    //cells    
    hipMemcpy((cuPtr->activeC), (carrays->activeC), ncells*sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy((cuPtr->actCells), (carrays->actCells), ncells*sizeof(int), hipMemcpyHostToDevice );    

    hipMemcpy((cuPtr->cidx), (carrays->cidx), ncells*sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy((cuPtr->nneig), (carrays->nneig), ncells*sizeof(int), hipMemcpyHostToDevice );

    hipMemcpy((cuPtr->z), (carrays->z), ncells*sizeof(double), hipMemcpyHostToDevice );
    hipMemcpy((cuPtr->h), (carrays->h), ncells*sizeof(double), hipMemcpyHostToDevice );
    hipMemcpy((cuPtr->hu), (carrays->hu), ncells*sizeof(double), hipMemcpyHostToDevice );
    hipMemcpy((cuPtr->hv), (carrays->hv), ncells*sizeof(double), hipMemcpyHostToDevice );
    hipMemcpy((cuPtr->u), (carrays->u), ncells*sizeof(double), hipMemcpyHostToDevice );
    hipMemcpy((cuPtr->v), (carrays->v), ncells*sizeof(double), hipMemcpyHostToDevice );
    hipMemcpy((cuPtr->modulou), (carrays->modulou), ncells*sizeof(double), hipMemcpyHostToDevice );
    hipMemcpy((cuPtr->sqrh), (carrays->sqrh), ncells*sizeof(double), hipMemcpyHostToDevice );
	
    hipMemcpy((cuPtr->area), (carrays->area), ncells*sizeof(double), hipMemcpyHostToDevice );
    hipMemcpy((cuPtr->nman), (carrays->nman), ncells*sizeof(double), hipMemcpyHostToDevice );
    hipMemcpy((cuPtr->SOX), (carrays->SOX), ncells*sizeof(double), hipMemcpyHostToDevice );
    hipMemcpy((cuPtr->SOY), (carrays->SOY), ncells*sizeof(double), hipMemcpyHostToDevice );    

    hipMemcpy((cuPtr->mass), (carrays->mass), ncells*sizeof(double), hipMemcpyHostToDevice );


    //cells*NCwall
    hipMemcpy((cuPtr->dh), (carrays->dh), nWallCell*sizeof(double), hipMemcpyHostToDevice );
    hipMemcpy((cuPtr->dhu), (carrays->dhu), nWallCell*sizeof(double), hipMemcpyHostToDevice );
    hipMemcpy((cuPtr->dhv), (carrays->dhv), nWallCell*sizeof(double), hipMemcpyHostToDevice );

    hipMemcpy((cuPtr->solidWallByCell), (carrays->solidWallByCell), nWallCell*sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy((cuPtr->neighCell), (carrays->neighCell), nWallCell*sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy((cuPtr->neighWall), (carrays->neighWall), nWallCell*sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy((cuPtr->typeWallByCell), (carrays->typeWallByCell), nWallCell*sizeof(int), hipMemcpyHostToDevice );

    hipMemcpy((cuPtr->normalXbyCell), (carrays->normalXbyCell), nWallCell*sizeof(double), hipMemcpyHostToDevice );
    hipMemcpy((cuPtr->normalYbyCell), (carrays->normalYbyCell), nWallCell*sizeof(double), hipMemcpyHostToDevice );    


    //internal walls
    hipMemcpy((cuPtr->activeW), (carrays->activeW), nwc*sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy((cuPtr->actWalls), (carrays->actWalls), nwc*sizeof(int), hipMemcpyHostToDevice );

    hipMemcpy((cuPtr->widx), (carrays->widx), nwc*sizeof(int), hipMemcpyHostToDevice );

    hipMemcpy((cuPtr->idx1), (carrays->idx1), nwc*sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy((cuPtr->idx2), (carrays->idx2), nwc*sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy((cuPtr->idw1), (carrays->idw1), nwc*sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy((cuPtr->idw2), (carrays->idw2), nwc*sizeof(int), hipMemcpyHostToDevice );

    hipMemcpy((cuPtr->normalX), (carrays->normalX), nwc*sizeof(double), hipMemcpyHostToDevice );
    hipMemcpy((cuPtr->normalY), (carrays->normalY), nwc*sizeof(double), hipMemcpyHostToDevice ); 
    hipMemcpy((cuPtr->deltaX), (carrays->deltaX), nwc*sizeof(double), hipMemcpyHostToDevice ); 
    hipMemcpy((cuPtr->length), (carrays->length), nwc*sizeof(double), hipMemcpyHostToDevice ); 
    hipMemcpy((cuPtr->distNormal), (carrays->distNormal), nwc*sizeof(double), hipMemcpyHostToDevice ); 
    hipMemcpy((cuPtr->distCentX), (carrays->distCentX), nwc*sizeof(double), hipMemcpyHostToDevice ); 
    hipMemcpy((cuPtr->distCentY), (carrays->distCentY), nwc*sizeof(double), hipMemcpyHostToDevice ); 

    hipMemcpy((cuPtr->nman2wall), (carrays->nman2wall), nwc*sizeof(double), hipMemcpyHostToDevice ); 
    hipMemcpy((cuPtr->gp), (carrays->gp), nwc*sizeof(double), hipMemcpyHostToDevice ); 

    hipMemcpy((cuPtr->typeOfBound), (carrays->typeOfBound), nwc*sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy((cuPtr->solidWall), (carrays->solidWall), nwc*sizeof(int), hipMemcpyHostToDevice );
 
    hipMemcpy((cuPtr->qnormalL), (carrays->qnormalL), nwc*sizeof(double), hipMemcpyHostToDevice ); 
    hipMemcpy((cuPtr->localDt), (carrays->localDt), nwc*sizeof(double), hipMemcpyHostToDevice ); 

    return(1);
    
}


////////////////////////////////////////////////////////////////
__global__ void assignMeshArraysToCudaMem(t_arrays *garrays,
	//------------------------cells
	int *activeC,
	int *actCells,
	int *cidx,
	int *nneig,
	double *z,
	double *h,
	double *hu,
	double *hv,
	double *u,
	double *v,
	double *modulou,
	double *sqrh,
	double *area,
	double *nman,
	double *SOX,
	double *SOY,
	double *mass,
	//--------------------- cells*NCwall
	double *dh,
	double *dhu,
	double *dhv,
	int *solidWallByCell,
	int *neighCell,
	int *neighWall,
    int *typeWallByCell,
	double *normalXbyCell,
	double *normalYbyCell,
	//---------------------- internal walls
	int *activeW,
	int *actWalls,
	int *widx,
	int *idx1,
	int *idx2,
	int *idw1,
	int *idw2,
	double *normalX,
	double *normalY,
	double *deltaX,
	double *length,
	double *distNormal,
	double *distCentX,
	double *distCentY,
	double *nman2wall,
	double *gp,
	int *typeOfBound,
	int *solidWall,
	double *qnormalL,
	double *localDt){
/*----------------------------*/
	//mesh
	garrays->activeC=activeC;
	garrays->actCells=actCells;

	garrays->cidx=cidx;
	garrays->nneig=nneig;

	garrays->z=z;
	garrays->h=h;
	garrays->hu=hu;
	garrays->hv=hv;
	garrays->u=u;
	garrays->v=v;
	garrays->modulou=modulou;
	garrays->sqrh=sqrh;

	garrays->area=area;
	garrays->nman=nman;
	garrays->SOX=SOX;
	garrays->SOY=SOY;

	garrays->mass=mass;

	//cells*NCwall
	garrays->dh=dh;
	garrays->dhu=dhu;
	garrays->dhv=dhv;

	garrays->solidWallByCell=solidWallByCell;
	garrays->neighCell=neighCell;
	garrays->neighWall=neighWall;
    garrays->typeWallByCell=typeWallByCell;

	garrays->normalXbyCell=normalXbyCell;
	garrays->normalYbyCell=normalYbyCell;

	//internal walls
	garrays->activeW=activeW;
	garrays->actWalls=actWalls;

	garrays->widx=widx;

	garrays->idx1=idx1;
	garrays->idx2=idx2;
	garrays->idw1=idw1;
	garrays->idw2=idw2;

	garrays->normalX=normalX;
	garrays->normalY=normalY;
	garrays->deltaX=deltaX;
	garrays->length=length;
	garrays->distNormal=distNormal;
	garrays->distCentX=distCentX;
	garrays->distCentY=distCentY;

	garrays->nman2wall=nman2wall;
	garrays->gp=gp;

	garrays->typeOfBound=typeOfBound;
	garrays->solidWall=solidWall;

	garrays->qnormalL=qnormalL;
	garrays->localDt=localDt;

}




////////////////////////////////////////////////////////////////
EXPORT_DLL int freeCudaMemory(t_cuPtr *cuPtr){
/*----------------------------*/

    //computation controls
    hipFree(cuPtr->index);
    hipFree(cuPtr->check);

    hipFree(cuPtr->t);
    hipFree(cuPtr->dt);

    hipFree(cuPtr->nActCells);
    hipFree(cuPtr->nActWalls);

    hipFree(cuPtr->nIter);
    hipFree(cuPtr->indexOut);
    hipFree(cuPtr->indexDump);
    hipFree(cuPtr->dumpComponent);
    hipFree(cuPtr->dumpState);

    hipFree(cuPtr->massOld);
    hipFree(cuPtr->massNew);
    hipFree(cuPtr->massError);


    //cells
    hipFree(cuPtr->activeC);
    hipFree(cuPtr->actCells);

    hipFree(cuPtr->cidx);
    hipFree(cuPtr->nneig);

    hipFree(cuPtr->z);
    hipFree(cuPtr->h);
    hipFree(cuPtr->hu);
    hipFree(cuPtr->hv);
    hipFree(cuPtr->u);
    hipFree(cuPtr->v);
    hipFree(cuPtr->modulou);
    hipFree(cuPtr->sqrh);

    hipFree(cuPtr->area);
    hipFree(cuPtr->nman);
    hipFree(cuPtr->SOX);
    hipFree(cuPtr->SOY);  

    hipFree(cuPtr->mass);


    //cells*NCwall
    hipFree(cuPtr->dh);
    hipFree(cuPtr->dhu);
    hipFree(cuPtr->dhv);

    hipFree(cuPtr->solidWallByCell);
    hipFree(cuPtr->neighCell);
    hipFree(cuPtr->neighWall);
    hipFree(cuPtr->typeWallByCell);

    hipFree(cuPtr->normalXbyCell);
    hipFree(cuPtr->normalYbyCell);


    //internal walls
    hipFree(cuPtr->activeW);
    hipFree(cuPtr->actWalls);

    hipFree(cuPtr->widx);

    hipFree(cuPtr->idx1);
    hipFree(cuPtr->idx2);
    hipFree(cuPtr->idw1);
    hipFree(cuPtr->idw2);

    hipFree(cuPtr->normalX);
    hipFree(cuPtr->normalY);
    hipFree(cuPtr->deltaX);
    hipFree(cuPtr->length);
    hipFree(cuPtr->distNormal);
    hipFree(cuPtr->distCentX);
    hipFree(cuPtr->distCentY);

    hipFree(cuPtr->nman2wall);
    hipFree(cuPtr->gp);

    hipFree(cuPtr->typeOfBound);
    hipFree(cuPtr->solidWall);

    hipFree(cuPtr->qnormalL);
    hipFree(cuPtr->localDt);
      
    return(1);

}





////////////////////////////////////////////////////////////////
EXPORT_DLL int allocateBoundArraysCudaMem(
    int nOBC, int nInlet, int nOutlet, 
    int nTotalBoundCells, int nTotalInnerCells,
    int nTotalPointSeries, 
    int nSolutes,
    t_cuPtr *cuPtr){
/*----------------------------*/


    if(nOBC){

        //bound geometry
        hipMalloc((void**) &(cuPtr->nCellsOBC), nOBC*sizeof(int));
        hipMalloc((void**) &(cuPtr->iniIndexOBC), nOBC*sizeof(int));
        hipMalloc((void**) &(cuPtr->idBoundOBC), nOBC*sizeof(int));
        hipMalloc((void**) &(cuPtr->typeOBC), nOBC*sizeof(int));
        hipMalloc((void**) &(cuPtr->flagInitializeOBC), nOBC*sizeof(int));

        hipMalloc((void**) &(cuPtr->blockSectionOBC), nOBC*sizeof(double));
        hipMalloc((void**) &(cuPtr->normalXOBC), nOBC*sizeof(double));
        hipMalloc((void**) &(cuPtr->normalYOBC), nOBC*sizeof(double));
        hipMalloc((void**) &(cuPtr->totalLengthOBC), nOBC*sizeof(double));
        hipMalloc((void**) &(cuPtr->totalAreaOBC), nOBC*sizeof(double));

        hipMalloc((void**) &(cuPtr->cellZminOBC), nOBC*sizeof(int));
        hipMalloc((void**) &(cuPtr->nInnerCellsOBC), nOBC*sizeof(int));
        hipMalloc((void**) &(cuPtr->iniInnerIndexOBC), nOBC*sizeof(int));

        //bound cells
        hipMalloc((void**) &(cuPtr->cidxBound), nTotalBoundCells*sizeof(int));
        hipMalloc((void**) &(cuPtr->zCellBound), nTotalBoundCells*sizeof(double));
        hipMalloc((void**) &(cuPtr->areaCellBound), nTotalBoundCells*sizeof(double));
        hipMalloc((void**) &(cuPtr->nxWallBound), nTotalBoundCells*sizeof(double));
        hipMalloc((void**) &(cuPtr->nyWallBound), nTotalBoundCells*sizeof(double));
        hipMalloc((void**) &(cuPtr->lWallBound), nTotalBoundCells*sizeof(double));

        // hipMalloc((void**) &(cuPtr->localh), nTotalBoundCells*sizeof(double));
        // hipMalloc((void**) &(cuPtr->localhu), nTotalBoundCells*sizeof(double));
        // hipMalloc((void**) &(cuPtr->localhv), nTotalBoundCells*sizeof(double));
        // #if SET_SOLUTE
        // hipMalloc((void**) &(cuPtr->localhphi), nSolutes*nTotalBoundCells*sizeof(double));
        // #endif
        
        //inner cells
        // if(nTotalInnerCells){
        //     hipMalloc((void**) &(cuPtr->cidxInner), nTotalInnerCells*sizeof(int));
        // }

        //time series
        hipMalloc((void**) &(cuPtr->nPointsSeriesOBC), nOBC*sizeof(int)); 
        hipMalloc((void**) &(cuPtr->iniIndexSeriesOBC), nOBC*sizeof(int));

        hipMalloc((void**) &(cuPtr->tSeriesOBC), nTotalPointSeries*sizeof(double));
        hipMalloc((void**) &(cuPtr->qSeriesOBC), nTotalPointSeries*sizeof(double));
        hipMalloc((void**) &(cuPtr->hzSeriesOBC), nTotalPointSeries*sizeof(double));
        hipMalloc((void**) &(cuPtr->frSeriesOBC), nTotalPointSeries*sizeof(double));

        #if SET_SOLUTE
        hipMalloc((void**) &(cuPtr->phiSeriesOBC), nSolutes*nTotalPointSeries*sizeof(double));
        #endif

        //mass balance pointers
        hipMalloc((void**) &(cuPtr->qBoundByCell), nTotalBoundCells*sizeof(double));
        hipMalloc((void**) &(cuPtr->mBoundByCell), nTotalBoundCells*sizeof(double));

        // if(nTotalInnerCells){
        //     hipMalloc((void**) &(cuPtr->mInnerByCell), nTotalInnerCells*sizeof(double));
        // }

        if(nInlet){
            hipMalloc((void**) &(cuPtr->qInByInlet), nInlet*sizeof(double));
            hipMalloc((void**) &(cuPtr->mInByInlet), nInlet*sizeof(double));
            hipMalloc((void**) &(cuPtr->aux1sByInlet), nInlet*sizeof(double));
        }
        if(nOutlet){
            hipMalloc((void**) &(cuPtr->qOutByOutlet), nOutlet*sizeof(double));
            hipMalloc((void**) &(cuPtr->mOutByOutlet), nOutlet*sizeof(double));
            hipMalloc((void**) &(cuPtr->aux1sByOutlet), nOutlet*sizeof(double));
        }

        hipMalloc((void**) &(cuPtr->qTotalIn), sizeof(double));
        hipMalloc((void**) &(cuPtr->qTotalOut), sizeof(double)); 
        hipMalloc((void**) &(cuPtr->mTotalIn), sizeof(double));
        hipMalloc((void**) &(cuPtr->mTotalOut), sizeof(double)); 

        hipMalloc((void**) &(cuPtr->massTotalIn), sizeof(double));
        hipMalloc((void**) &(cuPtr->massTotalOut), sizeof(double));

    }  

    return 1;    

}


////////////////////////////////////////////////////////////////
EXPORT_DLL int copyBoundSetupArraysCudaMem(
    t_arrays *carrays,
    t_arrays *garrays,
    t_cuPtr *cuPtr){
/*----------------------------*/

    int nOBC = carrays->nOBC;
    int nInlet = carrays->nInlet;
    int nOutlet = carrays->nOutlet;
    int nTotalPointSeries = carrays->nTotalPointSeries;
    int nSolutes = carrays->nSolutes;

    int i;
    double *aux1s;

    //ARRAYS ----------------------------------
    hipMemcpy(&(garrays->nTotalCellsIn), &(carrays->nTotalCellsIn), sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy(&(garrays->nTotalCellsOut), &(carrays->nTotalCellsOut), sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy(&(garrays->nTotalBoundCells), &(carrays->nTotalBoundCells), sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy(&(garrays->nTotalBoundCells), &(carrays->nTotalBoundCells), sizeof(int), hipMemcpyHostToDevice );

    hipMemcpy(&(garrays->nMaxBoundCells), &(carrays->nMaxBoundCells), sizeof(int), hipMemcpyHostToDevice );  
 
    hipMemcpy(&(garrays->nTotalPointSeries), &(carrays->nTotalPointSeries), sizeof(int), hipMemcpyHostToDevice );

    hipMemcpy(&(garrays->qTotalIn), &(carrays->qTotalIn), sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy(&(garrays->qTotalOut), &(carrays->qTotalOut), sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy(&(garrays->massTotalIn), &(carrays->massTotalIn), sizeof(int), hipMemcpyHostToDevice );  
    hipMemcpy(&(garrays->massTotalIn), &(carrays->massTotalIn), sizeof(int), hipMemcpyHostToDevice );
  
    // POINTERS ----------------------------------
    if(nOBC){

        //bound geometry
        hipMemcpy((cuPtr->nCellsOBC), (carrays->nCellsOBC), nOBC*sizeof(int), hipMemcpyHostToDevice );
        hipMemcpy((cuPtr->iniIndexOBC), (carrays->iniIndexOBC), nOBC*sizeof(int), hipMemcpyHostToDevice );
        hipMemcpy((cuPtr->idBoundOBC), (carrays->idBoundOBC), nOBC*sizeof(int), hipMemcpyHostToDevice );
        hipMemcpy((cuPtr->typeOBC), (carrays->typeOBC), nOBC*sizeof(int), hipMemcpyHostToDevice );
        hipMemcpy((cuPtr->flagInitializeOBC), (carrays->flagInitializeOBC), nOBC*sizeof(int), hipMemcpyHostToDevice );

        hipMemcpy((cuPtr->blockSectionOBC), (carrays->blockSectionOBC), nOBC*sizeof(double), hipMemcpyHostToDevice );
        hipMemcpy((cuPtr->normalXOBC), (carrays->normalXOBC), nOBC*sizeof(double), hipMemcpyHostToDevice );
        hipMemcpy((cuPtr->normalYOBC), (carrays->normalYOBC), nOBC*sizeof(double), hipMemcpyHostToDevice );
        hipMemcpy((cuPtr->totalLengthOBC), (carrays->totalLengthOBC), nOBC*sizeof(double), hipMemcpyHostToDevice );
        hipMemcpy((cuPtr->totalAreaOBC), (carrays->totalAreaOBC), nOBC*sizeof(double), hipMemcpyHostToDevice );

        hipMemcpy((cuPtr->cellZminOBC), (carrays->cellZminOBC), nOBC*sizeof(int), hipMemcpyHostToDevice );
        //hipMemcpy((cuPtr->nInnerCellsOBC), (carrays->nInnerCellsOBC), nOBC*sizeof(int), hipMemcpyHostToDevice );
        //hipMemcpy((cuPtr->iniInnerIndexOBC), (carrays->iniInnerIndexOBC), nOBC*sizeof(int), hipMemcpyHostToDevice );

        //time series
        hipMemcpy((cuPtr->nPointsSeriesOBC), (carrays->nPointsSeriesOBC), nOBC*sizeof(int), hipMemcpyHostToDevice );
        hipMemcpy((cuPtr->iniIndexSeriesOBC), (carrays->iniIndexSeriesOBC), nOBC*sizeof(int), hipMemcpyHostToDevice );
        
        hipMemcpy((cuPtr->tSeriesOBC), (carrays->tSeriesOBC), nTotalPointSeries*sizeof(double), hipMemcpyHostToDevice );
        hipMemcpy((cuPtr->qSeriesOBC), (carrays->qSeriesOBC), nTotalPointSeries*sizeof(double), hipMemcpyHostToDevice );
        hipMemcpy((cuPtr->hzSeriesOBC), (carrays->hzSeriesOBC), nTotalPointSeries*sizeof(double), hipMemcpyHostToDevice );
        hipMemcpy((cuPtr->frSeriesOBC), (carrays->frSeriesOBC), nTotalPointSeries*sizeof(double), hipMemcpyHostToDevice );

        #if SET_SOLUTE
        hipMemcpy((cuPtr->phiSeriesOBC), (carrays->phiSeriesOBC), nSolutes*nTotalPointSeries*sizeof(double), hipMemcpyHostToDevice );
        #endif

        //mass balance pointers  
        if(nInlet){
            hipMemcpy(cuPtr->qInByInlet, &(carrays->qInByInlet), nInlet*sizeof(double), hipMemcpyHostToDevice );
            hipMemcpy(cuPtr->mInByInlet, &(carrays->mInByInlet), nInlet*sizeof(double), hipMemcpyHostToDevice );

            aux1s = (double*)malloc(nInlet*sizeof(double));
            for(i=0;i<nInlet;i++){
                aux1s[i]=1.;
            }
            hipMemcpy(cuPtr->aux1sByInlet, aux1s, nInlet*sizeof(double), hipMemcpyHostToDevice );
            free(aux1s);
        }
        if(nOutlet){
            hipMemcpy(cuPtr->qOutByOutlet, &(carrays->qOutByOutlet), nOutlet*sizeof(double), hipMemcpyHostToDevice );      
            hipMemcpy(cuPtr->mOutByOutlet, &(carrays->mOutByOutlet), nOutlet*sizeof(double), hipMemcpyHostToDevice );  

            aux1s = (double*)malloc(nOutlet*sizeof(double));
            for(i=0;i<nOutlet;i++){
                aux1s[i]=1.;
            }
            hipMemcpy(cuPtr->aux1sByOutlet, aux1s, nOutlet*sizeof(double), hipMemcpyHostToDevice );
            free(aux1s);                
        }

        hipMemcpy(cuPtr->qTotalIn, &(carrays->qTotalIn), sizeof(double), hipMemcpyHostToDevice );
        hipMemcpy(cuPtr->qTotalOut, &(carrays->qTotalOut), sizeof(double), hipMemcpyHostToDevice );
        hipMemcpy(cuPtr->mTotalIn, &(carrays->mTotalIn), sizeof(double), hipMemcpyHostToDevice );
        hipMemcpy(cuPtr->mTotalOut, &(carrays->mTotalOut), sizeof(double), hipMemcpyHostToDevice );

        hipMemcpy(cuPtr->massTotalIn, &(carrays->massTotalIn), sizeof(double), hipMemcpyHostToDevice );
        hipMemcpy(cuPtr->massTotalOut, &(carrays->massTotalOut), sizeof(double), hipMemcpyHostToDevice );        
    }

    return 1;

}


////////////////////////////////////////////////////////////////
EXPORT_DLL int copyBoundMeshArraysCudaMem(
    t_arrays *carrays,
    t_arrays *garrays,
    t_cuPtr *cuPtr){
/*----------------------------*/

    int nOBC = carrays->nOBC;
    int nTotalBoundCells = carrays->nTotalBoundCells;
    int nTotalInnerCells = carrays->nTotalInnerCells;

    if(nOBC){
        //bound cells
        hipMemcpy((cuPtr->cidxBound), (carrays->cidxBound), nTotalBoundCells*sizeof(int), hipMemcpyHostToDevice );
        hipMemcpy((cuPtr->zCellBound), (carrays->zCellBound), nTotalBoundCells*sizeof(double), hipMemcpyHostToDevice );
        hipMemcpy((cuPtr->areaCellBound), (carrays->areaCellBound), nTotalBoundCells*sizeof(double), hipMemcpyHostToDevice );

        //bound walls
        hipMemcpy((cuPtr->nxWallBound), (carrays->nxWallBound), nTotalBoundCells*sizeof(double), hipMemcpyHostToDevice );
        hipMemcpy((cuPtr->nyWallBound), (carrays->nyWallBound), nTotalBoundCells*sizeof(double), hipMemcpyHostToDevice );
        hipMemcpy((cuPtr->lWallBound), (carrays->lWallBound), nTotalBoundCells*sizeof(double), hipMemcpyHostToDevice );

        //inner cells
        // if(nTotalInnerCells){
        //     hipMemcpy((cuPtr->cidxInner), (carrays->cidxInner), nTotalInnerCells*sizeof(int), hipMemcpyHostToDevice );
        // }

    }

    return 1;

}



////////////////////////////////////////////////////////////////
__global__ void assignBoundArraysToCudaMem(t_arrays *garrays,
	//------------------------ bound geometry
	int *nCellsOBC,
    int *iniIndexOBC,
    int *idBoundOBC,
    int *typeOBC,
    int *flagInitializeOBC,
    double *blockSectionOBC,
    double *normalXOBC,
    double *normalYOBC,
    double *totalLengthOBC,
    double *totalAreaOBC,
    int *cellZminOBC,
    int *nInnerCellsOBC,
    int *iniInnerIndexOBC,
    //----------------------- bound cells
    int *cidxBound,
    double *zCellBound,
    double *areaCellBound,
    double *nxWallBound,
    double *nyWallBound,
    double *lWallBound,
    //----------------------- inner cells
    int *cidxInner,
    //----------------------- time series
    int *nPointsSeriesOBC,
    int *iniIndexSeriesOBC,
    double *tSeriesOBC,
    double *qSeriesOBC,
    double *hzSeriesOBC,
    double *frSeriesOBC,
    double *phiSeriesOBC,
    //----------------------- mass balance pointers
    double *qBoundByCell,
    double *mBoundByCell,
    double *mInnerByCell,
    double *qInByInlet,
    double *qOutByOutlet,
    double *mInByInlet, 
    double *mOutByOutlet){
/*----------------------------*/

    if(garrays->nOBC){

        //bound geometry
        garrays->nCellsOBC=nCellsOBC;
        garrays->iniIndexOBC=iniIndexOBC;
        garrays->idBoundOBC=idBoundOBC;
        garrays->typeOBC=typeOBC;
        garrays->flagInitializeOBC=flagInitializeOBC;

        garrays->blockSectionOBC=blockSectionOBC;
        garrays->normalXOBC=normalXOBC;
        garrays->normalYOBC=normalYOBC;
        garrays->totalLengthOBC=totalLengthOBC;
        garrays->totalAreaOBC=totalAreaOBC;

        garrays->cellZminOBC=cellZminOBC;
        garrays->nInnerCellsOBC=nInnerCellsOBC;
        garrays->iniInnerIndexOBC=iniInnerIndexOBC;

        //bound cells
        garrays->cidxBound=cidxBound;
        garrays->zCellBound=zCellBound;
        garrays->areaCellBound=areaCellBound;
        garrays->nxWallBound=nxWallBound;
        garrays->nyWallBound=nyWallBound;
        garrays->lWallBound=lWallBound;  

        //inner cells 
        // if(garrays->nTotalInnerCells){
        //     garrays->cidxInner=cidxInner;
        // }     

        //time series
        garrays->nPointsSeriesOBC=nPointsSeriesOBC;
        garrays->iniIndexSeriesOBC=iniIndexSeriesOBC;

        garrays->tSeriesOBC=tSeriesOBC;
        garrays->qSeriesOBC=qSeriesOBC;
        garrays->hzSeriesOBC=hzSeriesOBC;
        garrays->frSeriesOBC=frSeriesOBC;

        #if SET_SOLUTE
        garrays->phiSeriesOBC=phiSeriesOBC;
        #endif

        //mass balance pointers
        garrays->qBoundByCell=qBoundByCell;
        garrays->mBoundByCell=mBoundByCell;
        // if(garrays->nTotalInnerCells){
        //     garrays->mInnerByCell=mInnerByCell;
        // }   

        if(garrays->nInlet){
            garrays->qInByInlet=qInByInlet;
            garrays->mInByInlet=mInByInlet;
        }
        if(garrays->nOutlet){
            garrays->qOutByOutlet=qOutByOutlet;
            garrays->mOutByOutlet=mOutByOutlet;
        }

    }

}


////////////////////////////////////////////////////////////////
EXPORT_DLL int freeBoundaCudaMemory(
    int nOBC, int nInlet, int nOutlet, 
    int nTotalBoundCells, int nTotalInnerCells,
    int nTotalPointSeries,
    t_cuPtr *cuPtr){
/*----------------------------*/

    if(nOBC){
        hipFree(cuPtr->nCellsOBC);
        hipFree(cuPtr->iniIndexOBC);
        hipFree(cuPtr->idBoundOBC);
        hipFree(cuPtr->typeOBC);
        hipFree(cuPtr->flagInitializeOBC);


        hipFree(cuPtr->blockSectionOBC);
        hipFree(cuPtr->normalXOBC);
        hipFree(cuPtr->normalYOBC);
        hipFree(cuPtr->totalLengthOBC);
        hipFree(cuPtr->totalAreaOBC);

        hipFree(cuPtr->cellZminOBC);
        hipFree(cuPtr->nInnerCellsOBC);
        hipFree(cuPtr->iniInnerIndexOBC);

        hipFree(cuPtr->cidxBound);
        hipFree(cuPtr->zCellBound);
        hipFree(cuPtr->areaCellBound);
        hipFree(cuPtr->nxWallBound);
        hipFree(cuPtr->nyWallBound);
        hipFree(cuPtr->lWallBound);

        if(nTotalInnerCells){
            hipFree(cuPtr->cidxInner);
        }

        hipFree(cuPtr->nPointsSeriesOBC);
        hipFree(cuPtr->iniIndexSeriesOBC);

        hipFree(cuPtr->tSeriesOBC);
        hipFree(cuPtr->qSeriesOBC);
        hipFree(cuPtr->hzSeriesOBC);
        hipFree(cuPtr->frSeriesOBC);


        #if SET_SOLUTE
        hipFree(cuPtr->phiSeriesOBC);
        #endif        

        hipFree(cuPtr->qBoundByCell);
        hipFree(cuPtr->mBoundByCell);

        if(nTotalInnerCells){
            hipFree(cuPtr->mInnerByCell);
        }

        if(nInlet){
            hipFree(cuPtr->qInByInlet);
            hipFree(cuPtr->mInByInlet);
            hipFree(cuPtr->aux1sByInlet);
        }
        if(nOutlet){
            hipFree(cuPtr->qOutByOutlet);
            hipFree(cuPtr->mOutByOutlet);
            hipFree(cuPtr->aux1sByOutlet);
        }

        hipFree(cuPtr->qTotalIn);
        hipFree(cuPtr->qTotalOut);
        hipFree(cuPtr->mTotalIn);
        hipFree(cuPtr->mTotalOut);

        hipFree(cuPtr->massTotalIn);
        hipFree(cuPtr->massTotalOut);         
    } 

    return(1);    

}





#if SET_SOLUTE
////////////////////////////////////////////////////////////////
EXPORT_DLL int allocateSoluteArraysCudaMem(
    int nSolutes, 
    int NCwall, int ncells, int nWallCell, int nwc, int nwb, 
    t_cuPtr *cuPtr){
/*----------------------------*/

    if(nSolutes){
        hipMalloc((void**) &(cuPtr->Dtd), sizeof(double));
        hipMalloc((void**) &(cuPtr->dtAux), sizeof(double));

        //solute control arrays
        hipMalloc((void**) &(cuPtr->typeDiff), nSolutes*sizeof(int));
        hipMalloc((void**) &(cuPtr->k_xx), nSolutes*sizeof(double));
        hipMalloc((void**) &(cuPtr->k_yy), nSolutes*sizeof(double));

        //cells
        hipMalloc((void**) &(cuPtr->hphi), nSolutes*ncells*sizeof(double));
        hipMalloc((void**) &(cuPtr->phi), nSolutes*ncells*sizeof(double));
        hipMalloc((void**) &(cuPtr->localDtd), nSolutes*ncells*sizeof(double));
        hipMalloc((void**) &(cuPtr->BTcell), nSolutes*ncells*sizeof(double));

        //cells*NCwall
        hipMalloc((void**) &(cuPtr->dhphi), nSolutes*nWallCell*sizeof(double));
        hipMalloc((void**) &(cuPtr->Bwall), nSolutes*nWallCell*sizeof(double));
    }

    return(1);

}



////////////////////////////////////////////////////////////////
int copySoluteArraysCudaMem(
    int nSolutes,
    t_arrays *carrays,
    t_arrays *garrays,
    t_cuPtr *cuPtr){
/*----------------------------*/
    
	int i,j;
    int NCwall;
	int ncells,nwc,nwb;
    int nWallCell;

	//Local variables just for allocation
    NCwall=carrays->NCwall;	//walls per cell
	ncells=carrays->ncells;
    nWallCell=NCwall*ncells;
	nwc=carrays->nw_calc;
	nwb=carrays->nw_bound;

    if(nSolutes){
        hipMemcpy(&(garrays->flagDiffusion), &(carrays->flagDiffusion), sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(cuPtr->Dtd, &(carrays->Dtd), sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(&(garrays->Dtd), &(carrays->Dtd), sizeof(double), hipMemcpyHostToDevice);

        //solutes
        hipMemcpy((cuPtr->typeDiff), (carrays->typeDiff), nSolutes*sizeof(int), hipMemcpyHostToDevice );
        hipMemcpy((cuPtr->k_xx), (carrays->k_xx), nSolutes*sizeof(double), hipMemcpyHostToDevice );
        hipMemcpy((cuPtr->k_yy), (carrays->k_yy), nSolutes*sizeof(double), hipMemcpyHostToDevice );        

        //solutes*cells    
        hipMemcpy((cuPtr->hphi), (carrays->hphi), nSolutes*ncells*sizeof(double), hipMemcpyHostToDevice );
        hipMemcpy((cuPtr->phi), (carrays->phi), nSolutes*ncells*sizeof(double), hipMemcpyHostToDevice );
        hipMemcpy((cuPtr->localDtd), (carrays->localDtd), nSolutes*ncells*sizeof(double), hipMemcpyHostToDevice );
        hipMemcpy((cuPtr->BTcell), (carrays->BTcell), nSolutes*ncells*sizeof(double), hipMemcpyHostToDevice );
    
        //solutes*cells*NCwall
        hipMemcpy((cuPtr->dhphi), (carrays->dhphi), nSolutes*nWallCell*sizeof(double), hipMemcpyHostToDevice );
        hipMemcpy((cuPtr->Bwall), (carrays->Bwall), nSolutes*nWallCell*sizeof(double), hipMemcpyHostToDevice );
    }

    return(1);

}  



////////////////////////////////////////////////////////////////
__global__ void assignSoluteArraysToCudaMem(int nSolutes, t_arrays *garrays,
	//------------------------cells
	int *typeDiff,
	double *k_xx,
	double *k_yy,
	double *hphi,
	double *phi,
    double *localDtd,
    double *BTcell,
	double *dhphi,
    double *Bwall){
/*----------------------------*/

    if(nSolutes){    
        //solute controls
        garrays->typeDiff=typeDiff;
        garrays->k_xx=k_xx;
        garrays->k_yy=k_yy;

        //mesh
        garrays->hphi=hphi;
        garrays->phi=phi;
        garrays->localDtd=localDtd;
        garrays->BTcell=BTcell;

        //cells*NCwall
        garrays->dhphi=dhphi;
        garrays->Bwall=Bwall;
    }

}


////////////////////////////////////////////////////////////////
EXPORT_DLL int freeSoluteCudaMemory(
    int nSolutes, 
    t_cuPtr *cuPtr){
/*----------------------------*/

    if(nSolutes){
        
        hipFree(cuPtr->Dtd);
        hipFree(cuPtr->dtAux);

        //solute control arrays
        hipFree(cuPtr->typeDiff);
        hipFree(cuPtr->k_xx);
        hipFree(cuPtr->k_yy);

        //cells
        hipFree(cuPtr->hphi);
        hipFree(cuPtr->phi);
        hipFree(cuPtr->localDtd);
        hipFree(cuPtr->BTcell);

        //cells*NCwall
        hipFree(cuPtr->dhphi);
        hipFree(cuPtr->Bwall);
    }

    return(1);

}

#endif
