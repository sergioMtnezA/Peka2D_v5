#include "hip/hip_runtime.h"
#include "calculate.cuh"

hipblasHandle_t cuHandle;

////////////////////////////////////////////////////////////////
EXPORT_DLL int computeSimulation(
    t_parameters spar,
    t_mesh *mesh, 
    t_arrays *carrays,
    t_timers *timers,  
	t_message *msg){
/*----------------------------*/

	FILE *fp;
	char temp[1024], filename[1024];

    double t;
    int nIter;

    clock_t stime1, stime2;
    clock_t start0, end0;

    t_arrays *garrays;
    t_cuPtr cuPtr; 
    int nTasks, blocksPerGrid;

    t=carrays->ti;
    nIter=carrays->nIter;
     

 	//Start GPU initialization time .....................................
	stime1=clock();

    //Select GPU device
    int num_devices, device;
    hipGetDeviceCount(&num_devices);
    sprintf(temp,"Num. GPU devices: %d",num_devices);
    Notify(temp,MSG_L0,msg);	

    hipDeviceProp_t prop;
    if(num_devices > 1){
        int max_multiprocessors = 0, max_device = 0;
        for (device = 0; device < num_devices; device++){
            hipGetDeviceProperties(&prop, device);
            if (max_multiprocessors < prop.multiProcessorCount) {
                max_multiprocessors = prop.multiProcessorCount;
                max_device = device;
            }
        }
        device=max_device;
    }else{
        device = 0;
    }
    hipSetDevice(device);
    sprintf(temp,"Selected device ID: %d",device);
    Notify(temp,MSG_L3,msg);

    hipGetDeviceProperties(&prop, device);
    sprintf(temp,"Compute capability %d.%d",prop.major, prop.minor);
    Notify(temp,MSG_L0,msg);     
    int coresPerSM = getCoresPerSM(prop.major, prop.minor);
    sprintf(temp,"CUDA sm %d || threads %d", prop.multiProcessorCount, prop.multiProcessorCount*coresPerSM);
    Notify(temp,MSG_L0,msg);  
    sprintf(temp,"Shared memory: Maximum %zu KB - Limit %zu KB",(prop.sharedMemPerBlock/1024),(prop.sharedMemPerBlockOptin/1024));
    Notify(temp,MSG_L0,msg);      


    //Initialize CUBLAS
	hipblasCreate(&cuHandle);
	hipblasSetPointerMode(cuHandle, HIPBLAS_POINTER_MODE_DEVICE);
    sprintf(temp,"CUBLAS methods initialized");
    Notify(temp,MSG_L3,msg);	


    //Set HEAP memory size for local malloc
    //size_t heapsize = 64*1024*1024; //64MB 
    //hipDeviceSetLimit(hipLimitMallocHeapSize, heapsize);     
    //sprintf(temp,"HEAP memory set");
    //Notify(temp,MSG_L3,msg);


    //create CUDA memory and initialize
    hipMalloc( (void **) &garrays, sizeof(t_arrays));
    if( createArraysCudaMemory(carrays, garrays, &(cuPtr), msg) ){
        sprintf(temp,"CUDA memory allocated and initialized");
        Notify(temp,MSG_L3,msg);	
    }

	stime2=clock();
	timers->initGPU += double(stime2-stime1)/CLOCKS_PER_SEC;
	//sprintf(temp,"Initializtion time %.6lf",timers->init);
	//Notify(temp,MSG_L0,msg);
	//End GPU initialization time .....................................      




 	//Start IO time .....................................
	stime1=clock();

    //initialize computation files
    if(create_computation_files(spar.dir, msg)){
        sprintf(temp,"Computation files created");
        Notify(temp,MSG_L2,msg);	
    } 


    //initialize boundary conditions
    if( computeInitialBoundaryConditions(carrays, garrays, &(cuPtr), msg) ){
        sprintf(temp,"Initial boundary conditions set");
        Notify(temp,MSG_L2,msg);	
    }     


    //initialize mass balance
    if( computeInitialMassBalance(carrays, garrays, &(cuPtr), msg) ){
        write_massBalance(spar.dir, carrays, msg);
        sprintf(temp,"Initial mass in domain %lf m3",carrays->massNew);
        Notify(temp,MSG_L2,msg);	
    }       


    //write initial condition      
    sprintf(filename,"%sstate%d.vtk",spar.dir,0);
    if(write_vtk_state(filename,mesh,carrays,msg)){
        sprintf(temp,"State%d VTK file written",0);
        Notify(temp,MSG_L0,msg);	
    } 

	stime2=clock();
	timers->init += double(stime2-stime1)/CLOCKS_PER_SEC;
	//sprintf(temp,"Initializtion time %.6lf",timers->init);
	//Notify(temp,MSG_L0,msg);
	//End IO time .....................................   




	/////////////////////////////////////////////////////////////////////////
	// TIME LOOP
	/////////////////////////////////////////////////////////////////////////
    sprintf(temp,"Simulation starts ...............");
    Notify(temp,MSG_L1,msg);

    //Start computation time .....................................
	start0=clock();

	while(t < carrays->tf){

        //update carrays to next time step
		generateTimeStep(&t, carrays, garrays, &(cuPtr), timers, msg);

     	//Start IO time .....................................
        stime1=clock();

        //print screen info
        if(carrays->nIter%carrays->nIterOut==0){
            dump_screen_info(carrays, msg);
        }

        //write components results
        if(carrays->dumpComponent){ 
            //mass balance file
            write_massBalance(spar.dir, carrays, msg);  

            carrays->indexDump++;	
        }        

        //write output results
        if(carrays->dumpState){
            //VTK file
            sprintf(filename,"%sstate%d.vtk",spar.dir,carrays->indexOut);
            if(write_vtk_state(filename,mesh,carrays,msg)){
                sprintf(temp,"State%d VTK file written",carrays->indexOut);
                Notify(temp,MSG_L1,msg);	
            } 

            carrays->indexOut++;	
        }

        carrays->nIter++;

        stime2=clock();
        timers->writeOut += double(stime2-stime1)/CLOCKS_PER_SEC;
        //End IO time .....................................          


        // sprintf(filename,"%sprueba.vtk",spar.dir);
        // if(write_vtk_state(filename,mesh,carrays,msg)){
        //     sprintf(temp,"Prueba VTK file written");
        //     Notify(temp,MSG_L1,msg);	
        // }  
        // getchar();


	}
	/////////////////////////////////////////////////////////////////////////
	// END TIME LOOP
	/////////////////////////////////////////////////////////////////////////        
       
	end0=clock();
	timers->computeSim += double(end0-start0)/CLOCKS_PER_SEC;
	//sprintf(temp,"Computation time %.6lf",timers->computeSim);
	//Notify(temp,MSG_L0,msg);
	//End computation time .....................................

    sprintf(temp,"Computation loop completed");
    Notify(temp,MSG_L1,msg);



    //Start closing time .....................................
	start0=clock();

    //write hotstart initialization file      
    sprintf(filename,"%shotstart.out",spar.dir);
    if(write_hotstart_file(filename,carrays,msg)){
		sprintf(temp,"Hotstart initialization file written");
		Notify(temp,MSG_L1,msg);        
    }    

    // Free CUDA memory
    freeBoundaCudaMemory(carrays->nOBC, carrays->nInlet, carrays->nOutlet,
        carrays->nTotalBoundCells, carrays->nTotalInnerCells, 
        carrays->nTotalPointSeries,
        &(cuPtr));
    freeCudaMemory(&(cuPtr));
    hipFree(garrays);
    hipblasDestroy(cuHandle);

    end0=clock();
	timers->closeSim += double(end0-start0)/CLOCKS_PER_SEC;
    //End closing time .....................................

	return 1;
}




////////////////////////////////////////////////////////////////
EXPORT_DLL int computeInitialBoundaryConditions(
    t_arrays *carrays,
    t_arrays *garrays,     
    t_cuPtr *cuPtr,
    t_message *msg){
/*----------------------------*/

    int i;
    int icount, ocount;

    int nTasks, blocksPerGrid;
    int obcPerGrid;
    size_t memPerOBC;    

    char temp[1024];

    //Initialize open boundaries
    carrays->qTotalIn=0.0;
    carrays->qTotalOut=0.0;
    hipMemset(&(garrays->qTotalOut), 0, sizeof(double));
    hipMemset(&(garrays->qTotalOut), 0, sizeof(double));

    carrays->mTotalIn=0.0;
    carrays->mTotalOut=0.0;
    hipMemset(&(garrays->mTotalOut), 0, sizeof(double));
    hipMemset(&(garrays->mTotalOut), 0, sizeof(double));   

    if(carrays->nOBC){

        nTasks=carrays->nTotalBoundCells;
        obcPerGrid = carrays->nOBC; 
        memPerOBC = 4*carrays->nMaxBoundCells*sizeof(double);
        sprintf(temp,"Shared memory: nMaxBoundCells %d - Reserved %zu KB",carrays->nMaxBoundCells,(memPerOBC/1024));
        Notify(temp,MSG_L0,msg);

        hipFuncSetCacheConfig(reinterpret_cast<const void*>(g_update_open_boundary), hipFuncCachePreferShared);
        g_update_open_boundary <<<obcPerGrid,threadsPerOBC,memPerOBC>>> (nTasks, garrays, 
            cuPtr->qBoundByCell, cuPtr->mBoundByCell, cuPtr->mInnerByCell,
            cuPtr->qInByInlet,cuPtr->mInByInlet,
            cuPtr->qOutByOutlet,cuPtr->mOutByOutlet);          


        //-------------------------------------------------------------
        //hipblasDasum(cuHandle, carrays->nInlet, cuPtr->mInByInlet, 1, cuPtr->mTotalIn);
        hipblasDdot(cuHandle, carrays->nInlet, cuPtr->aux1sByInlet, 1, cuPtr->mInByInlet, 1, cuPtr->mTotalIn);
        hipMemcpy(&(garrays->mTotalIn), cuPtr->mTotalIn, sizeof(double), hipMemcpyDeviceToDevice );
        hipMemcpy(&(carrays->mTotalIn), cuPtr->mTotalIn, sizeof(double), hipMemcpyDeviceToHost );

        //hipblasDasum(cuHandle, carrays->nOutlet, cuPtr->mOutByOutlet, 1, cuPtr->mTotalOut);
        hipblasDdot(cuHandle, carrays->nOutlet, cuPtr->aux1sByOutlet, 1, cuPtr->mOutByOutlet, 1, cuPtr->mTotalOut);
        hipMemcpy(&(garrays->mTotalOut), cuPtr->mTotalOut, sizeof(double), hipMemcpyDeviceToDevice );
        hipMemcpy(&(carrays->mTotalOut), cuPtr->mTotalOut, sizeof(double), hipMemcpyDeviceToHost );

        sprintf(temp,"Initial added mass: Inlet %lf m3 - Outlet %lf m3",carrays->mTotalIn, carrays->mTotalOut);
        Notify(temp,MSG_L2,msg);   

        //-------------------------------------------------------------------
        //hipblasDasum(cuHandle, carrays->nInlet, cuPtr->qInByInlet, 1, cuPtr->qTotalIn);
        hipblasDdot(cuHandle, carrays->nInlet, cuPtr->aux1sByInlet, 1, cuPtr->qInByInlet, 1, cuPtr->qTotalIn);
        hipMemcpy(&(garrays->qTotalIn), cuPtr->qTotalIn, sizeof(double), hipMemcpyDeviceToDevice );
        hipMemcpy(&(carrays->qTotalIn), cuPtr->qTotalIn, sizeof(double), hipMemcpyDeviceToHost );

        //hipblasDasum(cuHandle, carrays->nOutlet, cuPtr->qOutByOutlet, 1, cuPtr->qTotalOut);
        hipblasDdot(cuHandle, carrays->nOutlet, cuPtr->aux1sByOutlet, 1, cuPtr->qOutByOutlet, 1, cuPtr->qTotalOut);        
        hipMemcpy(&(garrays->qTotalOut), cuPtr->qTotalOut, sizeof(double), hipMemcpyDeviceToDevice );
        hipMemcpy(&(carrays->qTotalOut), cuPtr->qTotalOut, sizeof(double), hipMemcpyDeviceToHost );

        sprintf(temp,"Initial discharge: Inlet %lf m3/s - Outlet %lf m3/s",carrays->qTotalIn, carrays->qTotalOut);
        Notify(temp,MSG_L2,msg);          
    }    

    //Inititalize mass balance
    carrays->massTotalIn=0.0;
    carrays->massTotalOut=0.0;
    hipMemset(&(garrays->massTotalIn), 0, sizeof(double));
    hipMemset(&(garrays->massTotalOut), 0, sizeof(double));    

    return 1;
}




////////////////////////////////////////////////////////////////
EXPORT_DLL int computeInitialMassBalance(
    t_arrays *carrays,
    t_arrays *garrays,     
    t_cuPtr *cuPtr,
    t_message *msg){
/*----------------------------*/

    int nTasks, blocksPerGrid;

    nTasks=carrays->nActCells;
    blocksPerGrid = nTasks/threadsPerBlock + 1; 
    g_compute_cell_mass <<<blocksPerGrid,threadsPerBlock>>> (nTasks, garrays, cuPtr->mass);
    
    hipblasDasum(cuHandle, carrays->ncells, cuPtr->mass, 1, cuPtr->massNew);
    hipMemcpy(&(garrays->massNew), cuPtr->massNew, sizeof(double), hipMemcpyDeviceToDevice );
    hipMemcpy(&(carrays->massNew), cuPtr->massNew, sizeof(double), hipMemcpyDeviceToHost );

    return 1;
}






////////////////////////////////////////////////////////////////
EXPORT_DLL void generateTimeStep(
    double *t,
    t_arrays *carrays,
    t_arrays *garrays,     
    t_cuPtr *cuPtr,
    t_timers *timers, 
    t_message *msg){
/*----------------------------*/

    int i;
	int checkpos;
    int ncells=carrays->ncells;
    int nwc=carrays->nw_calc;
    int nSteps;
    double dtDifR, dtAux;

    int icount, ocount;

    int nTasks, blocksPerGrid;
    int obcPerGrid;
    size_t memPerOBC;

    clock_t stime1, stime2;

    carrays->massOld = carrays->massNew;
    hipMemcpy(&(garrays->massOld), &(garrays->massNew), sizeof(double), hipMemcpyDeviceToDevice );

    hipMemcpy(&(garrays->nIter), &(carrays->nIter), sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy(&(garrays->indexOut), &(carrays->indexOut), sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy(&(garrays->indexDump), &(carrays->indexDump), sizeof(int), hipMemcpyHostToDevice );



    //Start wallCalculus time .....................................
	stime1=clock();

    nTasks=carrays->nWallCell;
    blocksPerGrid = nTasks/threadsPerBlock + 1; 
	g_initialize_delta <<<blocksPerGrid,threadsPerBlock>>> (nTasks, garrays);


    nTasks=carrays->nActWalls;
    blocksPerGrid = nTasks/threadsPerBlock + 1; 
    g_wall_rotated_calculus <<<blocksPerGrid,threadsPerBlock>>> (nTasks, garrays, cuPtr->localDt);
    hipMemcpy(&(carrays->nActCells), &(garrays->nActCells), sizeof(int), hipMemcpyDeviceToHost );
  
    
    nTasks=carrays->nw_calc;
    hipblasIdamin(cuHandle, nTasks, cuPtr->localDt, 1, cuPtr->index);
	g_get_dtmin <<<1,1>>>  (garrays, cuPtr->localDt, cuPtr->index);


    nTasks=carrays->nActCells;
    blocksPerGrid = nTasks/threadsPerBlock + 1; 
    g_update_contributions <<<blocksPerGrid,threadsPerBlock>>> (nTasks, garrays);


    // Sincronizar la CPU con la GPU
    hipDeviceSynchronize();

	stime2=clock();
	timers->wallCalculus += double(stime2-stime1)/CLOCKS_PER_SEC;
    //End wallCalculus time .....................................  



    //Start cellUpdating time .....................................
	stime1=clock();

    checkpos=1;
    while(checkpos){
        hipMemset(cuPtr->check, 0, sizeof(int)); //set the integer to 0
        //displayCudaIscalar <<<1,1>>> (cuPtr->check);

        nTasks=carrays->nActCells;
        blocksPerGrid = nTasks/threadsPerBlock + 1; 
        g_checkpos_h <<<blocksPerGrid,threadsPerBlock>>> (nTasks, garrays, cuPtr->check);
        hipMemcpy(&(checkpos), cuPtr->check, sizeof(int), hipMemcpyDeviceToHost );

        if(checkpos){
            // printf("checkpos %d\n",checkpos);
            g_reduce_dt <<<1,1>>> (garrays);
        }
    }


    g_set_new_dt <<<1,1>>> (garrays);
    hipMemcpy(&(carrays->t), &(garrays->t), sizeof(double), hipMemcpyDeviceToHost );
    hipMemcpy(&(carrays->dt), &(garrays->dt), sizeof(double), hipMemcpyDeviceToHost );
    hipMemcpy(&(carrays->dumpComponent), &(garrays->dumpComponent), sizeof(int), hipMemcpyDeviceToHost );
    hipMemcpy(&(carrays->dumpState), &(garrays->dumpState), sizeof(int), hipMemcpyDeviceToHost );
    (*t) = carrays->t;


    nTasks=carrays->nActCells;
    blocksPerGrid = nTasks/threadsPerBlock + 1; 
	g_update_cells <<<blocksPerGrid,threadsPerBlock>>> (nTasks, garrays);
    hipMemcpy(&(carrays->nActWalls), &(garrays->nActWalls), sizeof(int), hipMemcpyDeviceToHost );

    // Sincronizar la CPU con la GPU
    hipDeviceSynchronize();

    stime2=clock();
	timers->cellUpdating += double(stime2-stime1)/CLOCKS_PER_SEC;
    //End cellUpdating time .....................................  


    //Start wetDryFix time .....................................
	stime1=clock();

    nTasks=carrays->nActWalls;
    blocksPerGrid = nTasks/threadsPerBlock + 1; 
	g_check_wetdry <<<blocksPerGrid,threadsPerBlock>>> (nTasks, garrays);
    
    
    nTasks=carrays->nActCells;
    blocksPerGrid = nTasks/threadsPerBlock + 1; 
	g_update_wetdry_cells <<<blocksPerGrid,threadsPerBlock>>> (nTasks, garrays);  

    // Sincronizar la CPU con la GPU
    hipDeviceSynchronize();

    stime2=clock();
	timers->wetDryFix += double(stime2-stime1)/CLOCKS_PER_SEC;
    //End wetDryFix time .....................................    



    //Start openBoundaries time .....................................   
    stime1=clock();
    if(carrays->nOBC){
        carrays->qTotalIn=0.0;
        carrays->qTotalOut=0.0;
        hipMemset(&(garrays->qTotalIn), 0, sizeof(double));
        hipMemset(&(garrays->qTotalOut), 0, sizeof(double));

        //hipblasDasum(cuHandle, carrays->nInlet, cuPtr->qInByInlet, 1, cuPtr->qTotalIn);
        hipblasDdot(cuHandle, carrays->nInlet, cuPtr->aux1sByInlet, 1, cuPtr->qInByInlet, 1, cuPtr->qTotalIn);
        hipMemcpy(&(garrays->qTotalIn), cuPtr->qTotalIn, sizeof(double), hipMemcpyDeviceToDevice );
        hipMemcpy(&(carrays->qTotalIn), cuPtr->qTotalIn, sizeof(double), hipMemcpyDeviceToHost );
         
        //hipblasDasum(cuHandle, carrays->nOutlet, cuPtr->qOutByOutlet, 1, cuPtr->qTotalOut);
        hipblasDdot(cuHandle, carrays->nOutlet, cuPtr->aux1sByOutlet, 1, cuPtr->qOutByOutlet, 1, cuPtr->qTotalOut);
        hipMemcpy(&(garrays->qTotalOut), cuPtr->qTotalOut, sizeof(double), hipMemcpyDeviceToDevice );
        hipMemcpy(&(carrays->qTotalOut), cuPtr->qTotalOut, sizeof(double), hipMemcpyDeviceToHost );


        //update boundaries
        hipMemset(cuPtr->qInByInlet, 0, carrays->nInlet*sizeof(double));
        hipMemset(cuPtr->mInByInlet, 0, carrays->nInlet*sizeof(double));
        hipMemset(cuPtr->qOutByOutlet, 0, carrays->nOutlet*sizeof(double));
        hipMemset(cuPtr->mOutByOutlet, 0, carrays->nOutlet*sizeof(double)); 
        
        nTasks=carrays->nTotalBoundCells;
        obcPerGrid = carrays->nOBC; 
        memPerOBC = 4*carrays->nMaxBoundCells*sizeof(double);
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(g_update_open_boundary), hipFuncCachePreferShared);
        g_update_open_boundary <<<obcPerGrid,threadsPerOBC,memPerOBC>>> (nTasks, garrays, 
            cuPtr->qBoundByCell, cuPtr->mBoundByCell, cuPtr->mInnerByCell,
            cuPtr->qInByInlet,cuPtr->mInByInlet,
            cuPtr->qOutByOutlet,cuPtr->mOutByOutlet);


        carrays->mTotalIn=0.0;
        carrays->mTotalOut=0.0;
        hipMemset(&(garrays->mTotalIn), 0, sizeof(double));
        hipMemset(&(garrays->mTotalOut), 0, sizeof(double));        

        //hipblasDasum(cuHandle, carrays->nInlet, cuPtr->mInByInlet, 1, cuPtr->mTotalIn);
        hipblasDdot(cuHandle, carrays->nInlet, cuPtr->aux1sByInlet, 1, cuPtr->mInByInlet, 1, cuPtr->mTotalIn);
        hipMemcpy(&(garrays->mTotalIn), cuPtr->mTotalIn, sizeof(double), hipMemcpyDeviceToDevice );
        hipMemcpy(&(carrays->mTotalIn), cuPtr->mTotalIn, sizeof(double), hipMemcpyDeviceToHost ); 
            

        //hipblasDasum(cuHandle, carrays->nOutlet, cuPtr->mOutByOutlet, 1, cuPtr->mTotalOut);
        hipblasDdot(cuHandle, carrays->nOutlet, cuPtr->aux1sByOutlet, 1, cuPtr->mOutByOutlet, 1, cuPtr->mTotalOut);
        hipMemcpy(&(garrays->mTotalOut), cuPtr->mTotalOut, sizeof(double), hipMemcpyDeviceToDevice );     
        hipMemcpy(&(carrays->mTotalOut), cuPtr->mTotalOut, sizeof(double), hipMemcpyDeviceToHost ); 
  
    }

    // Sincronizar la CPU con la GPU
    hipDeviceSynchronize();
    stime2=clock();
	timers->boundConditon += double(stime2-stime1)/CLOCKS_PER_SEC;   
    //End openBoundaries time .....................................   




    //Start memoryTransfer time .....................................
	stime1=clock();

    nTasks=carrays->nActCells;
    blocksPerGrid = nTasks/threadsPerBlock + 1; 
    g_compute_cell_mass <<<blocksPerGrid,threadsPerBlock>>> (nTasks, garrays, cuPtr->mass);    
    hipblasDasum(cuHandle, carrays->ncells, cuPtr->mass, 1, cuPtr->massNew);
    hipMemcpy(&(garrays->massNew), cuPtr->massNew, sizeof(double), hipMemcpyDeviceToDevice );
    hipMemcpy(&(carrays->massNew), cuPtr->massNew, sizeof(double), hipMemcpyDeviceToHost );

    g_compute_mass_error <<<1,1>>> (garrays);
    hipMemcpy(&(carrays->massError), &(garrays->massError), sizeof(double), hipMemcpyDeviceToHost );

    if(carrays->dumpComponent){
        // Transfer massBalance from GPU to CPU 
        hipMemcpy(&(carrays->qTotalIn), &(garrays->qTotalIn), sizeof(double), hipMemcpyDeviceToHost );
        hipMemcpy(&(carrays->qTotalOut), &(garrays->qTotalOut), sizeof(double), hipMemcpyDeviceToHost );
        hipMemcpy(&(carrays->mTotalIn), &(garrays->mTotalIn), sizeof(double), hipMemcpyDeviceToHost );
        hipMemcpy(&(carrays->mTotalOut), &(garrays->mTotalOut), sizeof(double), hipMemcpyDeviceToHost );                
        hipMemcpy(&(carrays->massTotalIn), &(garrays->massTotalIn), sizeof(double), hipMemcpyDeviceToHost );
        hipMemcpy(&(carrays->massTotalOut), &(garrays->massTotalOut), sizeof(double), hipMemcpyDeviceToHost );
    }

    if(carrays->dumpState){
        // Transfer flow arrays from GPU to CPU 
        //hipMemcpy((carrays->z), (cuPtr->z), ncells*sizeof(double), hipMemcpyDeviceToHost );
        hipMemcpy((carrays->h), (cuPtr->h), ncells*sizeof(double), hipMemcpyDeviceToHost );
        //hipMemcpy((carrays->hu), (cuPtr->hu), ncells*sizeof(double), hipMemcpyDeviceToHost );
        //hipMemcpy((carrays->hv), (cuPtr->hv), ncells*sizeof(double), hipMemcpyDeviceToHost );
        hipMemcpy((carrays->u), (cuPtr->u), ncells*sizeof(double), hipMemcpyDeviceToHost );
        hipMemcpy((carrays->v), (cuPtr->v), ncells*sizeof(double), hipMemcpyDeviceToHost );
        hipMemcpy((carrays->modulou), (cuPtr->modulou), ncells*sizeof(double), hipMemcpyDeviceToHost );          
    }

    // Sincronizar la CPU con la GPU
    hipDeviceSynchronize();
    stime2=clock();
	timers->memoryTransfer += double(stime2-stime1)/CLOCKS_PER_SEC;
    //End memoryTransfer time ..................................... 




    // Reconstruct actCells and actWalls arrays 
    #if RECONSTRUC_ACTIVE
    //Start wetDryFix time .....................................
	stime1=clock();

    //if(carrays->dumpState){
    if(carrays->nIter%nIterArrangeActElem==0){

        hipMemset(&(garrays->nActCells), 0, sizeof(int)); //set the integer to 0
        hipMemset((cuPtr->activeC), 0, ncells*sizeof(int)); //set the array to 0
        hipMemset((cuPtr->actCells), 0xFF, ncells*sizeof(int)); //set the array to -1

        hipMemset(&(garrays->nActWalls), 0, sizeof(int)); //set the integer to 0
        hipMemset((cuPtr->activeW), 0, nwc*sizeof(int)); //set the array to 0
        hipMemset((cuPtr->actWalls), 0xFF, nwc*sizeof(int));  //set the array to -1

        nTasks=carrays->nw_calc;
        blocksPerGrid = nTasks/threadsPerBlock + 1; 
        g_reconstruct_active_elements <<<blocksPerGrid,threadsPerBlock>>> (nTasks, garrays);
        hipMemcpy(&(carrays->nActWalls), &(garrays->nActWalls), sizeof(int), hipMemcpyDeviceToHost );
        //hipMemcpy((carrays->actWalls), (cuPtr->actWalls), nwc*sizeof(int), hipMemcpyDeviceToHost );
        hipMemcpy(&(carrays->nActCells), &(garrays->nActCells), sizeof(int), hipMemcpyDeviceToHost );
        //hipMemcpy((carrays->actCells), (cuPtr->actCells), ncells*sizeof(int), hipMemcpyDeviceToHost );
           
    }

    // Sincronizar la CPU con la GPU
    hipDeviceSynchronize();

    stime2=clock();
	timers->wetDryFix += double(stime2-stime1)/CLOCKS_PER_SEC;
    //End wetDryFix time .....................................  
    #endif


}

